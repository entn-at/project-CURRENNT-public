#include "hip/hip_runtime.h"
/******************************************************************************
 * This file is an addtional component of CURRENNT. 
 * Xin WANG
 * National Institute of Informatics, Japan
 * 2016
 *
 * This file is part of CURRENNT. 
 * Copyright (c) 2013 Johannes Bergmann, Felix Weninger, Bjoern Schuller
 * Institute for Human-Machine Communication
 * Technische Universitaet Muenchen (TUM)
 * D-80290 Munich, Germany
 *
 *
 * CURRENNT is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CURRENNT is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with CURRENNT.  If not, see <http://www.gnu.org/licenses/>.
 *****************************************************************************//*

*/


#ifdef _MSC_VER
#   pragma warning (disable: 4244) // thrust/iterator/iterator_adaptor.h(121): warning C4244: '+=' : conversion from '__int64' to 'int', possible loss of data
#endif

#include "FilteringLayer.hpp"

#include "../helpers/getRawPointer.cuh"
#include "../helpers/Matrix.hpp"
#include "../helpers/min.cuh"
#include "../helpers/max.cuh"
#include "../helpers/safeExp.cuh"
#include "../helpers/NumericLimits.cuh"
#include "../helpers/JsonClasses.hpp"
#include "../helpers/misFuncs.hpp"

#include "../activation_functions/Tanh.cuh"
#include "../activation_functions/Logistic.cuh"
#include "../activation_functions/Identity.cuh"
#include "../activation_functions/Relu.cuh"

#include "../Configuration.hpp"

#include <boost/foreach.hpp>
#include <boost/random/normal_distribution.hpp>
#include <boost/random/uniform_real_distribution.hpp>
#include <boost/random/mersenne_twister.hpp>
#include <boost/algorithm/string.hpp>
#include <boost/lexical_cast.hpp>

#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#include <sstream>
#include <fstream>
#include <cmath>


#define FILTERING_LAYER_MODE_NONE_SELECTIVE 0
#define FILTERING_LAYER_MODE_SELECTIVE 1

namespace internal{
namespace {

    // Use one group of filters, do filtering
    struct causalFilteringForward_none_selective
    {
	int        filterLength;
	int        filterShareAcrossDim;
	int        layerSize;
	int        parallel;
	
	real_t     *inputData;
	real_t     *filterCoeffs;
	const char *patTypes;   

	// From 1 : T (of the previous layer)
	__host__ __device__ void operator() (const thrust::tuple<real_t&, int> &t) const
	{
	    int outputIdx  = t.get<1>();
	    int dimIdx     = outputIdx % layerSize;  // dimension index
	    int timeIdx    = outputIdx / layerSize;  // time index (regardless of parallel)
	    
	    int BlockIdx   = timeIdx / parallel;     // time index (considering parallel mode)
	    int BlockInIdx = timeIdx % parallel;     // index within a parallel block


	    if (patTypes[timeIdx] == PATTYPE_NONE){
		t.get<0>() = 0;
		return;
	    }		

	    // accumulate and update
	    real_t tmp = 0.0;
	    real_t filterCoeff;
	    for (int idx = 0 ; idx < filterLength; idx++){

		// get the filter coefficient for one step
		// a_0 + a_1 z^-1 + ... + a_N z^-N
		// [a_0, a_1, ..., a_N]
		if (filterShareAcrossDim)
		    filterCoeff = filterCoeffs[idx];
		else
		    filterCoeff = filterCoeffs[dimIdx * filterLength + idx];
		
		if ((BlockIdx - idx) >= 0){
		    tmp += (inputData[((BlockIdx - idx) * parallel + BlockInIdx) * layerSize + dimIdx]
			    * filterCoeff);
		}
	    }
	    t.get<0>() = tmp;
	    
	}
    };
    
    
    struct causalFilteringBackward_none_selective
    {
	int        filterLength;
	int        filterShareAcrossDim;
	int        layerSize;
	int        maxLength;
	int        parallel;
	real_t     *inputErrors;
	real_t     *filterCoeffs;
	const char *patTypes;   

	// From 1 : T (of the previous layer)
	__host__ __device__ void operator() (const thrust::tuple<real_t&, int> &t) const
	{
	    int outputIdx  = t.get<1>();
	    int dimIdx     = outputIdx % layerSize;  // dimension index
	    int timeIdx    = outputIdx / layerSize;  // time index (regardless of parallel)
	    
	    int BlockIdx   = timeIdx / parallel;     // time index (considering parallel mode)
	    int BlockInIdx = timeIdx % parallel;     // index within a parallel block


	    if (patTypes[timeIdx] == PATTYPE_NONE){
		t.get<0>() = 0;
		return;
	    }		

	    // accumulate and update
	    real_t tmp = 0.0;
	    real_t filterCoeff;
	    for (int idx = 0 ; idx < filterLength; idx++){

		// get the filter coefficient for one step
		// a_0 + a_1 z^-1 + ... + a_N z^-N
		// [a_0, a_1, ..., a_N]
		if (filterShareAcrossDim)
		    filterCoeff = filterCoeffs[idx];
		else
		    filterCoeff = filterCoeffs[dimIdx * filterLength + idx];
		
		if (((BlockIdx + idx) * parallel + BlockInIdx) < maxLength &&
		    patTypes[((BlockIdx + idx) * parallel + BlockInIdx)] != PATTYPE_NONE){
		    tmp += (inputErrors[((BlockIdx+idx)*parallel+BlockInIdx) * layerSize + dimIdx]
			    * filterCoeff);
		}
	    }
	    t.get<0>() = tmp;
	    
	}
    };


    // Use weighted sum of filters
    struct causalFilteringForward_selective
    {
	int        filterLength;
	int        filterShareAcrossDim;
	
	int        outputLayerSize;
	int        inputLayerSize;
	int        filterNum;
	int        parallel;
	
	real_t     *inputData;
	real_t     *filterCoeffs;
	const char *patTypes;   

	// From 1 : T (of the previous layer)
	__host__ __device__ void operator() (const thrust::tuple<real_t&, int> &t) const
	{
	    int outputIdx  = t.get<1>();
	    int dimIdx     = outputIdx % outputLayerSize;  // dimension index
	    int timeIdx    = outputIdx / outputLayerSize;  // time index (regardless of parallel)
	    
	    int BlockIdx   = timeIdx / parallel;     // time index (considering parallel mode)
	    int BlockInIdx = timeIdx % parallel;     // index within a parallel block

	    if (patTypes[timeIdx] == PATTYPE_NONE){
		t.get<0>() = 0;
		return;
	    }		

	    // accumulate and update
	    real_t tmp = 0.0;
	    real_t filterCoeff = 0.0;
	    real_t filterWeight= 0.0;
	    
	    for (int idx = 0 ; idx < filterLength; idx++){

		// The input vector looks like:
		// [data_dim1, data_dim2, ..., data_dim_N, filter_w1, filter_w2, ...filter_wM]
		// M + N = inputLayerSize
		// N = outputLayerSize
		// M = filterNum
		
		filterCoeff = 0.0;
		
		// get weighted sum of filter weights
		for (int filterIdx = 0 ; filterIdx < filterNum; filterIdx++){

		    // weight of current time step
		    filterWeight = inputData[timeIdx * inputLayerSize +
					     inputLayerSize - filterNum + filterIdx];

		    // weighted sum of filter coeffs
		    if (filterShareAcrossDim)
			filterCoeff += (filterCoeffs[filterIdx * filterLength + idx] * filterWeight);
		    else
			filterCoeff += (filterCoeffs[(filterIdx * outputLayerSize + dimIdx) *
						     filterLength + idx] * filterWeight);
		}
		
		// filtering
		if ((BlockIdx - idx) >= 0){
		    tmp += (inputData[((BlockIdx-idx)*parallel+BlockInIdx)*inputLayerSize+dimIdx]
			    * filterCoeff);
		}
	    }
	    t.get<0>() = tmp;
	    
	}
    };
    
    
    struct causalFilteringBackward_selective
    {
	int        filterLength;
	int        filterShareAcrossDim;

	int        outputLayerSize;
	int        inputLayerSize;
	int        filterNum;


	int        maxLength;
	int        parallel;
	real_t     *inputData;
	real_t     *inputErrors;
	real_t     *filterCoeffs;
	const char *patTypes;   

	// From 1 : T (of the previous layer)
	__host__ __device__ void operator() (const thrust::tuple<real_t&, int> &t) const
	{
	    int outputIdx  = t.get<1>();
	    int dimIdx     = outputIdx % inputLayerSize;  // dimension index
	    int timeIdx    = outputIdx / inputLayerSize;  // time index (regardless of parallel)
	    
	    int BlockIdx   = timeIdx / parallel;     // time index (considering parallel mode)
	    int BlockInIdx = timeIdx % parallel;     // index within a parallel block


	    if (patTypes[timeIdx] == PATTYPE_NONE){
		t.get<0>() = 0;
		return;
	    }		

	    // accumulate and update
	    real_t tmp = 0.0;
	    real_t filterCoeff  = 0.0;
	    real_t filterWeight = 0.0;

	    if (dimIdx < (inputLayerSize - filterNum)){
		// gradients w.r.t input signal
		for (int idx = 0 ; idx < filterLength; idx++){
		    
		    // if the output time step is valid
		    if (((BlockIdx + idx) * parallel + BlockInIdx) < maxLength &&
			patTypes[((BlockIdx + idx) * parallel + BlockInIdx)] != PATTYPE_NONE){

			filterCoeff = 0.0;
			
			for (int filter_idx = 0 ; filter_idx < filterNum; filter_idx++){

			    // weight of that (output) time step 
			    filterWeight = inputData[((BlockIdx+idx) * parallel + BlockInIdx) *
						     inputLayerSize +
						     inputLayerSize-filterNum+filter_idx];
			    if (filterShareAcrossDim)
				filterCoeff += (filterCoeffs[filter_idx * filterLength + idx] *
						filterWeight);
			    else
				filterCoeff += (filterCoeffs[(filter_idx*outputLayerSize+dimIdx) *
							     filterLength + idx] *
						filterWeight);
			}
		
			// sum gradient
			tmp += (inputErrors[((BlockIdx + idx) * parallel + BlockInIdx) *
					    outputLayerSize + dimIdx]
				* filterCoeff);
		    }
		}
		t.get<0>() = tmp;
		
	    }else{

		// gradients w.r.t filter weights
		int filter_idx = dimIdx - outputLayerSize;

		// gradients w.r.t input signal
		for (int idx = 0 ; idx < filterLength; idx++){

		    if ((BlockIdx - idx) < 0)
			continue;
			    
		    if (filterShareAcrossDim){
			filterCoeff = filterCoeffs[filter_idx * filterLength + idx];
			for (int featDimIdx = 0 ; featDimIdx < outputLayerSize; featDimIdx ++){
			    tmp += (inputData[((BlockIdx - idx) * parallel + BlockInIdx) *
					      inputLayerSize + featDimIdx]
				    * filterCoeff
				    * inputErrors[(BlockIdx * parallel + BlockInIdx) *
						  outputLayerSize + featDimIdx]);
			}
			
		    }else{
			for (int featDimIdx = 0 ; featDimIdx < outputLayerSize; featDimIdx ++){
			    filterCoeff = filterCoeffs[(filter_idx * outputLayerSize + featDimIdx) *
						       filterLength + idx];
			    tmp += (inputData[((BlockIdx - idx) * parallel + BlockInIdx) *
					      inputLayerSize + featDimIdx]
				    * filterCoeff
				    * inputErrors[(BlockIdx * parallel + BlockInIdx) *
						  outputLayerSize + featDimIdx]);
			}
		    }
		}
		t.get<0>() = tmp;
	    }
	    
	}
    };
    
    
}
}

namespace layers {
    template <typename TDevice>
    FilteringLayer<TDevice>::FilteringLayer(const helpers::JsonValue &layerChild,
					    const helpers::JsonValue &weightsSection,
					    Layer<TDevice>           &precedingLayer,
					    int                       maxSeqLength,
					    int                       layerID)
	: TrainableLayer<TDevice>(layerChild, weightsSection, 0, 0,
				  precedingLayer, maxSeqLength, layerID)
    {
	m_filter_across_dim = ((layerChild->HasMember("shareAcrossDim")) ? 
			       ((*layerChild)["shareAcrossDim"].GetInt()) : 1);
	m_filter_coeffs_str = ((layerChild->HasMember("filterCoeffs")) ? 
			       ((*layerChild)["filterCoeffs"].GetString()) : "");
	m_filter_length = ((layerChild->HasMember("filterLength")) ? 
			       ((*layerChild)["filterLength"].GetInt()) : 0);

	if (this->size() == this->precedingLayer().size()){
	    // only 1 group of filter
	    m_filter_mode = FILTERING_LAYER_MODE_NONE_SELECTIVE;
	    m_filter_num = 1; 
	}else{
	    // multiplt groups of filters
	    m_filter_mode = FILTERING_LAYER_MODE_SELECTIVE;
	    // assume input signal contains the weights of each filter
	    m_filter_num = this->precedingLayer().size() - this->size();
	}
	
	
	if (this->getResolution() != this->precedingLayer().getResolution())
	    throw std::runtime_error("Error in filter layer: resolution != previous layer resolu");
	
	// parse the filter coefficients
	if (m_filter_coeffs_str.size()){
	    m_filter_coeffs.clear();
	    misFuncs::ParseFloatOpt(m_filter_coeffs_str, m_filter_coeffs_H);
	    m_filter_coeffs = m_filter_coeffs_H;

	    // check, when shareAcrossDim is False, #coefficients should be N * feature dimension
	    if (m_filter_across_dim == 0){
		if (m_filter_coeffs_H.size() % this->size() != 0){
		    printf("\n\t %d filter coefficients for %d dimensions,", this->size(),
			   (int)m_filter_coeffs_H.size());
		    throw std::runtime_error("Error in filtering layer: filterCoeffs invalid");
		}
	    }
	}else{
	    throw std::runtime_error("Error in filtering layer: no filterCoeffs");
	}

	
	int tmp_filter_length = 0;
	// if m_filter_length is not specified, infer it
	if (m_filter_across_dim == 0)
	    tmp_filter_length = m_filter_coeffs_H.size() / this->size() / m_filter_num;
	else
	    tmp_filter_length = m_filter_coeffs_H.size() / m_filter_num;

	
	if (m_filter_length == 0){
	    m_filter_length = tmp_filter_length;
	}else{
	    if (m_filter_length != tmp_filter_length)
		throw std::runtime_error("Error in filtering layer: filter mismatch");
	}

	// print information
	if (m_filter_mode == FILTERING_LAYER_MODE_NONE_SELECTIVE){
	    printf(" fixed filter, ");
	}else{
	    printf(" soft-weighted %d filters, ", m_filter_num);
	}
	if (m_filter_across_dim)
	    printf(" one filter (length %d) across feature dimension", m_filter_length);
	else
	    printf(" one filter (length %d) for each feature dimension", m_filter_length);

    }

    template <typename TDevice>
    FilteringLayer<TDevice>::~FilteringLayer()
    {
    }

    template <typename TDevice>
    const std::string& FilteringLayer<TDevice>::type() const
    {
        static std::string s;
        if (s.empty()) s = "filtering";
        return s;
    }

    template <typename TDevice>
    void FilteringLayer<TDevice>::computeForwardPass(const int nnState)
    {
	int timeLength = this->curMaxSeqLength() * this->parallelSequences();
	
	if (m_filter_mode == FILTERING_LAYER_MODE_NONE_SELECTIVE){
	    // one group of filters
	    internal::causalFilteringForward_none_selective fn1;
	    fn1.filterLength = this->m_filter_length;
	    fn1.layerSize    = this->size();
	    fn1.parallel     = this->parallelSequences();
	    fn1.filterShareAcrossDim = this->m_filter_across_dim;
	    fn1.filterCoeffs  = helpers::getRawPointer(this->m_filter_coeffs);
	    fn1.inputData = helpers::getRawPointer(this->precedingLayer().outputs());
	    fn1.patTypes  = helpers::getRawPointer(this->patTypes());
	    
	    int n = timeLength * this->size();
	    thrust::for_each(
               thrust::make_zip_iterator(
		  thrust::make_tuple(this->outputs().begin(),
				     thrust::counting_iterator<int>(0))),
	       thrust::make_zip_iterator(
		  thrust::make_tuple(this->outputs().begin()           + n,
				     thrust::counting_iterator<int>(0) + n)),
	       fn1);

	}else if (m_filter_mode == FILTERING_LAYER_MODE_SELECTIVE){
	    // weighted sum of filters
	    internal::causalFilteringForward_selective fn1;
	    fn1.filterLength    = this->m_filter_length;
	    fn1.outputLayerSize = this->size();
	    fn1.inputLayerSize  = this->precedingLayer().size();
	    fn1.filterNum       = m_filter_num;
	    fn1.filterLength    = m_filter_length;
	    fn1.parallel        = this->parallelSequences();
	    
	    fn1.filterShareAcrossDim = this->m_filter_across_dim;
	    
	    fn1.filterCoeffs  = helpers::getRawPointer(this->m_filter_coeffs);
	    fn1.inputData = helpers::getRawPointer(this->precedingLayer().outputs());
	    fn1.patTypes  = helpers::getRawPointer(this->patTypes());
	    
	    int n = timeLength * this->size();
	    thrust::for_each(
               thrust::make_zip_iterator(
		  thrust::make_tuple(this->outputs().begin(),
				     thrust::counting_iterator<int>(0))),
	       thrust::make_zip_iterator(
		  thrust::make_tuple(this->outputs().begin()           + n,
				     thrust::counting_iterator<int>(0) + n)),
	       fn1);

	}else{
	    throw std::runtime_error("Error in filtering layer: Unknown filter mode");
	}
    }

    template <typename TDevice>
    void FilteringLayer<TDevice>::computeForwardPass(const int timeStep, const int nnState)
    {
	throw std::runtime_error("Filtering computeForwardPass(timeStep) not implemented");
    }

    template <typename TDevice>
    void FilteringLayer<TDevice>::computeBackwardPass(const int nnState)
    {
	int timeLength = this->curMaxSeqLength() * this->parallelSequences();
	
	if (m_filter_mode == FILTERING_LAYER_MODE_NONE_SELECTIVE){    
	    internal::causalFilteringBackward_none_selective fn1;
	    fn1.filterLength = this->m_filter_length;
	    fn1.layerSize    = this->size();
	    fn1.maxLength    = timeLength;
	    fn1.filterShareAcrossDim = this->m_filter_across_dim;
	    fn1.parallel     = this->parallelSequences();
	    
	    fn1.filterCoeffs  = helpers::getRawPointer(this->m_filter_coeffs);
	    fn1.inputErrors = helpers::getRawPointer(this->outputErrors());
	    fn1.patTypes  = helpers::getRawPointer(this->patTypes());
	    
	    int n = timeLength * this->size();
	    thrust::for_each(
              thrust::make_zip_iterator(
		thrust::make_tuple(this->precedingLayer().outputErrors().begin(),
				   thrust::counting_iterator<int>(0))),
	      thrust::make_zip_iterator(
		thrust::make_tuple(this->precedingLayer().outputErrors().begin() + n,
				   thrust::counting_iterator<int>(0) + n)),
	      fn1);

	}else if (m_filter_mode == FILTERING_LAYER_MODE_SELECTIVE){
	    
	    internal::causalFilteringBackward_selective fn1;
	    fn1.filterLength     = this->m_filter_length;
	    fn1.outputLayerSize  = this->size();
	    fn1.inputLayerSize   = this->precedingLayer().size();
	    fn1.filterNum        = this->m_filter_num;
	    fn1.maxLength        = timeLength;
	    fn1.parallel         = this->parallelSequences();

	    fn1.filterShareAcrossDim = this->m_filter_across_dim;

	    fn1.filterCoeffs     = helpers::getRawPointer(this->m_filter_coeffs);
	    fn1.inputData        = helpers::getRawPointer(this->precedingLayer().outputs());
	    fn1.inputErrors      = helpers::getRawPointer(this->outputErrors());
	    fn1.patTypes         = helpers::getRawPointer(this->patTypes());
	    
	    int n = timeLength * this->precedingLayer().size();
	    thrust::for_each(
              thrust::make_zip_iterator(
		thrust::make_tuple(this->precedingLayer().outputErrors().begin(),
				   thrust::counting_iterator<int>(0))),
	      thrust::make_zip_iterator(
		thrust::make_tuple(this->precedingLayer().outputErrors().begin() + n,
				   thrust::counting_iterator<int>(0) + n)),
	      fn1);

	    
	}else{
	    throw std::runtime_error("Error in filtering layer: Unknown filter mode");
	}
	
    }
    
    template <typename TDevice>
    void FilteringLayer<TDevice>::computeBackwardPass(const int timeStep, const int nnState)
    {
	throw std::runtime_error("FilteringLayer computeBackwardPass(timeStep) not implemented");
    }
    
    template <typename TDevice>
    void FilteringLayer<TDevice>::exportLayer(
	const helpers::JsonValue     &layersArray, 
	const helpers::JsonAllocator &allocator) const
    {
	TrainableLayer<TDevice>::exportLayer(layersArray, allocator);
        (*layersArray)[layersArray->Size() - 1].AddMember("shareAcrossDim", m_filter_across_dim,
							  allocator);
	(*layersArray)[layersArray->Size() - 1].AddMember("filterCoeffs", m_filter_coeffs_str.c_str(),
							  allocator);
	
    }

    template <typename TDevice>
    void FilteringLayer<TDevice>::reduceOutputBuffer()
    {
	this->resizeOutputBuffer(this->parallelSequences() * this->size());	
	this->setSaveMemoryFlag(true);
	printf("\t[mem saved]");
    }
    
    template <typename TDevice>
    int FilteringLayer<TDevice>::outputBufPtrBias(const int timeStepTimesParallel,
						  const int nnState)
    {
	if (this->getSaveMemoryFlag()){
	    return timeStepTimesParallel * this->size();
	}else{
	    return 0;
	}
    }	

    template <typename TDevice>
    void FilteringLayer<TDevice>::clearAllBuffers()
    {
	this->clearOutputBuffer();
    }

    template <typename TDevice>
    void FilteringLayer<TDevice>::resizeAllBuffers(const int timeLength)
    {
	this->resizeOutputBuffer(timeLength * this->size());
    }

    template class FilteringLayer<Cpu>;
    template class FilteringLayer<Gpu>;
}
