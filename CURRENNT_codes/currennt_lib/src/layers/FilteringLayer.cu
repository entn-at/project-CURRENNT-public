#include "hip/hip_runtime.h"
/******************************************************************************
 * This file is an addtional component of CURRENNT. 
 * Xin WANG
 * National Institute of Informatics, Japan
 * 2016
 *
 * This file is part of CURRENNT. 
 * Copyright (c) 2013 Johannes Bergmann, Felix Weninger, Bjoern Schuller
 * Institute for Human-Machine Communication
 * Technische Universitaet Muenchen (TUM)
 * D-80290 Munich, Germany
 *
 *
 * CURRENNT is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CURRENNT is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with CURRENNT.  If not, see <http://www.gnu.org/licenses/>.
 *****************************************************************************//*

*/


#ifdef _MSC_VER
#   pragma warning (disable: 4244) // thrust/iterator/iterator_adaptor.h(121): warning C4244: '+=' : conversion from '__int64' to 'int', possible loss of data
#endif

#include "FilteringLayer.hpp"

#include "../helpers/getRawPointer.cuh"
#include "../helpers/Matrix.hpp"
#include "../helpers/min.cuh"
#include "../helpers/max.cuh"
#include "../helpers/safeExp.cuh"
#include "../helpers/NumericLimits.cuh"
#include "../helpers/JsonClasses.hpp"
#include "../helpers/misFuncs.hpp"

#include "../activation_functions/Tanh.cuh"
#include "../activation_functions/Logistic.cuh"
#include "../activation_functions/Identity.cuh"
#include "../activation_functions/Relu.cuh"

#include "../Configuration.hpp"

#include <boost/foreach.hpp>
#include <boost/random/normal_distribution.hpp>
#include <boost/random/uniform_real_distribution.hpp>
#include <boost/random/mersenne_twister.hpp>
#include <boost/algorithm/string.hpp>
#include <boost/lexical_cast.hpp>

#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#include <sstream>
#include <fstream>
#include <cmath>


namespace internal{
namespace {

    struct causalFilteringForward
    {
	int        filterLength;
	int        filterShareAcrossDim;
	int        layerSize;
	int        parallel;
	
	real_t     *inputData;
	real_t     *filterCoeffs;
	const char *patTypes;   

	// From 1 : T (of the previous layer)
	__host__ __device__ void operator() (const thrust::tuple<real_t&, int> &t) const
	{
	    int outputIdx  = t.get<1>();
	    int dimIdx     = outputIdx % layerSize;  // dimension index
	    int timeIdx    = outputIdx / layerSize;  // time index (regardless of parallel)
	    
	    int BlockIdx   = timeIdx / parallel;     // time index (considering parallel mode)
	    int BlockInIdx = timeIdx % parallel;     // index within a parallel block


	    if (patTypes[timeIdx] == PATTYPE_NONE){
		t.get<0>() = 0;
		return;
	    }		

	    // accumulate and update
	    real_t tmp = 0.0;
	    real_t filterCoeff;
	    for (int idx = 0 ; idx < filterLength; idx++){

		// get the filter coefficient for one step
		// a_0 + a_1 z^-1 + ... + a_N z^-N
		// [a_0, a_1, ..., a_N]
		if (filterShareAcrossDim)
		    filterCoeff = filterCoeffs[idx];
		else
		    filterCoeff = filterCoeffs[layerSize * filterLength + idx];
		
		if ((BlockIdx - idx) >= 0){
		    tmp += (inputData[((BlockIdx - idx) * parallel + BlockInIdx) * layerSize + dimIdx]
			    * filterCoeff);
		}
	    }
	    t.get<0>() = tmp;
	    
	}
    };
    

    struct causalFilteringBackward
    {
	int        filterLength;
	int        filterShareAcrossDim;
	int        layerSize;
	int        maxLength;
	int        parallel;
	real_t     *inputErrors;
	real_t     *filterCoeffs;
	const char *patTypes;   

	// From 1 : T (of the previous layer)
	__host__ __device__ void operator() (const thrust::tuple<real_t&, int> &t) const
	{
	    int outputIdx  = t.get<1>();
	    int dimIdx     = outputIdx % layerSize;  // dimension index
	    int timeIdx    = outputIdx / layerSize;  // time index (regardless of parallel)
	    
	    int BlockIdx   = timeIdx / parallel;     // time index (considering parallel mode)
	    int BlockInIdx = timeIdx % parallel;     // index within a parallel block


	    if (patTypes[timeIdx] == PATTYPE_NONE){
		t.get<0>() = 0;
		return;
	    }		

	    // accumulate and update
	    real_t tmp = 0.0;
	    real_t filterCoeff;
	    for (int idx = 0 ; idx < filterLength; idx++){

		// get the filter coefficient for one step
		// a_0 + a_1 z^-1 + ... + a_N z^-N
		// [a_0, a_1, ..., a_N]
		if (filterShareAcrossDim)
		    filterCoeff = filterCoeffs[idx];
		else
		    filterCoeff = filterCoeffs[layerSize * filterLength + idx];
		
		if (((BlockIdx + idx) * parallel + BlockInIdx) < maxLength &&
		    patTypes[((BlockIdx + idx) * parallel + BlockInIdx)] != PATTYPE_NONE){
		    tmp += (inputErrors[((BlockIdx+idx)*parallel+BlockInIdx) * layerSize + dimIdx]
			    * filterCoeff);
		}
	    }
	    t.get<0>() = tmp;
	    
	}
    };
    
}
}

namespace layers {
    template <typename TDevice>
    FilteringLayer<TDevice>::FilteringLayer(const helpers::JsonValue &layerChild,
					    const helpers::JsonValue &weightsSection,
					    Layer<TDevice>           &precedingLayer,
					    int                       maxSeqLength,
					    int                       layerID)
	: TrainableLayer<TDevice>(layerChild, weightsSection, 0, 0,
				  precedingLayer, maxSeqLength, layerID)
    {
	m_filter_causal = ((layerChild->HasMember("causal")) ?
			   ((*layerChild)["causal"].GetInt()) : 1);
	m_filter_across_dim = ((layerChild->HasMember("shareAcrossDim")) ? 
			       ((*layerChild)["shareAcrossDim"].GetInt()) : 1);
	m_filter_coeffs_str = ((layerChild->HasMember("filterCoeffs")) ? 
			       ((*layerChild)["filterCoeffs"].GetString()) : "");

	// check
	if (this->size() != this->precedingLayer().size())
	    throw std::runtime_error("Error in filter layer: size != previous layer size");
	if (this->getResolution() != this->precedingLayer().getResolution())
	    throw std::runtime_error("Error in filter layer: resolution != previous layer resolu");
	
	// parse the filter coefficients
	if (m_filter_coeffs_str.size()){
	    m_filter_coeffs.clear();
	    misFuncs::ParseFloatOpt(m_filter_coeffs_str, m_filter_coeffs_H);
	    m_filter_coeffs = m_filter_coeffs_H;

	    // check, when shareAcrossDim is False, #coefficients should be N * feature dimension
	    if (m_filter_across_dim == 0){
		if (m_filter_coeffs_H.size() % this->size() != 0){
		    printf("\n\t %d filter coefficients for %d dimensions,", this->size(),
			   (int)m_filter_coeffs_H.size());
		    throw std::runtime_error("Error in filtering layer: filterCoeffs invalid");
		}
		m_filter_length = m_filter_coeffs_H.size() / this->size();
	    }else{
		m_filter_length = m_filter_coeffs_H.size();
	    }
	}else{
	    throw std::runtime_error("Error in filtering layer: no filterCoeffs");
	}

	if (m_filter_causal)
	    printf("\n\tFiltering: causal,");
	else
	    printf("\n\tFiltering: non-causal,");
	if (m_filter_across_dim)
	    printf(" one filter (length %d) across feature dimension", m_filter_length);
	else
	    printf(" one filter (length %d) for each feature dimension", m_filter_length);
	
    }

    template <typename TDevice>
    FilteringLayer<TDevice>::~FilteringLayer()
    {
    }

    template <typename TDevice>
    const std::string& FilteringLayer<TDevice>::type() const
    {
        static std::string s;
        if (s.empty()) s = "filtering";
        return s;
    }

    template <typename TDevice>
    void FilteringLayer<TDevice>::computeForwardPass(const int nnState)
    {
	int timeLength = this->curMaxSeqLength() * this->parallelSequences();
	if (m_filter_causal){
	    
	    internal::causalFilteringForward fn1;
	    fn1.filterLength = this->m_filter_length;
	    fn1.layerSize    = this->size();
	    fn1.parallel     = this->parallelSequences();
	    fn1.filterShareAcrossDim = this->m_filter_across_dim;
	    fn1.filterCoeffs  = helpers::getRawPointer(this->m_filter_coeffs);
	    fn1.inputData = helpers::getRawPointer(this->precedingLayer().outputs());
	    fn1.patTypes  = helpers::getRawPointer(this->patTypes());
	    
	    int n = timeLength * this->size();
	    thrust::for_each(
               thrust::make_zip_iterator(
		  thrust::make_tuple(this->outputs().begin(),
				     thrust::counting_iterator<int>(0))),
	       thrust::make_zip_iterator(
		  thrust::make_tuple(this->outputs().begin()           + n,
				     thrust::counting_iterator<int>(0) + n)),
	       fn1);

	}else{
	    throw std::runtime_error("Not yet implemented for non causal filter");
	}
    }

    template <typename TDevice>
    void FilteringLayer<TDevice>::computeForwardPass(const int timeStep, const int nnState)
    {
	throw std::runtime_error("Not implemented");
    }

    template <typename TDevice>
    void FilteringLayer<TDevice>::computeBackwardPass(const int nnState)
    {
	int timeLength = this->curMaxSeqLength() * this->parallelSequences();
	
	if (m_filter_causal){    
	    internal::causalFilteringBackward fn1;
	    fn1.filterLength = this->m_filter_length;
	    fn1.layerSize    = this->size();
	    fn1.maxLength    = timeLength;
	    fn1.filterShareAcrossDim = this->m_filter_across_dim;
	    fn1.parallel     = this->parallelSequences();
	    
	    fn1.filterCoeffs  = helpers::getRawPointer(this->m_filter_coeffs);
	    fn1.inputErrors = helpers::getRawPointer(this->outputErrors());
	    fn1.patTypes  = helpers::getRawPointer(this->patTypes());
	    
	    int n = timeLength * this->size();
	    thrust::for_each(
              thrust::make_zip_iterator(
		thrust::make_tuple(this->precedingLayer().outputErrors().begin(),
				   thrust::counting_iterator<int>(0))),
	      thrust::make_zip_iterator(
		thrust::make_tuple(this->precedingLayer().outputErrors().begin() + n,
				   thrust::counting_iterator<int>(0) + n)),
	      fn1);

	}else{
	    throw std::runtime_error("Not yet implemented for non causal filter");
	}
	
    }
    
    
    template <typename TDevice>
    void FilteringLayer<TDevice>::exportLayer(
	const helpers::JsonValue     &layersArray, 
	const helpers::JsonAllocator &allocator) const
    {
	TrainableLayer<TDevice>::exportLayer(layersArray, allocator);
	(*layersArray)[layersArray->Size() - 1].AddMember("causal", m_filter_causal, allocator);
        (*layersArray)[layersArray->Size() - 1].AddMember("shareAcrossDim", m_filter_across_dim,
							  allocator);
	(*layersArray)[layersArray->Size() - 1].AddMember("filterCoeffs", m_filter_coeffs_str.c_str(),
							  allocator);
	
    }

    template <typename TDevice>
    void FilteringLayer<TDevice>::reduceOutputBuffer()
    {
	this->resizeOutputBuffer(this->parallelSequences() * this->size());	
	this->setSaveMemoryFlag(true);
	printf("\t[mem saved]");
    }
    
    template <typename TDevice>
    int FilteringLayer<TDevice>::outputBufPtrBias(const int timeStepTimesParallel,
						  const int nnState)
    {
	if (this->getSaveMemoryFlag()){
	    return timeStepTimesParallel * this->size();
	}else{
	    return 0;
	}
    }	

    template <typename TDevice>
    void FilteringLayer<TDevice>::clearAllBuffers()
    {
	this->clearOutputBuffer();
    }

    template <typename TDevice>
    void FilteringLayer<TDevice>::resizeAllBuffers(const int timeLength)
    {
	this->resizeOutputBuffer(timeLength * this->size());
    }

    template class FilteringLayer<Cpu>;
    template class FilteringLayer<Gpu>;
}
