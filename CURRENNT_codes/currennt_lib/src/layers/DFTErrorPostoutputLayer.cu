#include "hip/hip_runtime.h"
/******************************************************************************
 * This file is an addtional component of CURRENNT. 
 * Xin WANG
 * National Institute of Informatics, Japan
 * 2016 - 2019
 *
 * This file is part of CURRENNT.
 *
 * CURRENNT is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CURRENNT is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with CURRENNT.  If not, see <http://www.gnu.org/licenses/>.
 *****************************************************************************/


#include "DFTErrorPostoutputLayer.hpp"
#include "../helpers/getRawPointer.cuh"
#include "../helpers/Matrix.hpp"
#include "../helpers/FFTMat.hpp"
#include "../helpers/sigProcess.hpp"
#include "../helpers/JsonClasses.hpp"
#include "../helpers/misFuncs.hpp"

#include "../Configuration.hpp"
#include "../MacroDefine.hpp"

#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/for_each.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/fill.h>
#include <boost/foreach.hpp>
#include <boost/shared_ptr.hpp>
#include <boost/algorithm/string.hpp>
#include <vector>


// Configuration for HNM training (obsolete)
#define DFTERRORPOST_HNM_MODEL_1 1
#define DFTERRORPOST_HNM_MODEL_2 2
#define DFTERRORUV       10
#define DFTERRORUVSEARCH 2
#define DFTERROR_PI      3.141215

// Configuration for Multi-dimensional signal analysis
#define DFTMODEFORMULTIDIMSIGNAL_NONE    0
#define DFTMODEFORMULTIDIMSIGNAL_CONCATE 1

// Configuration for waveform pre-emphasis & de-emphasis
#define DFTPREEMPHASISCOEFF 0.94 // emphasis factor

namespace internal{
namespace{

    struct ComputeMseWaveform
    {
        int layerSize;

        const char *patTypes;

        __host__ __device__ real_t operator() (const thrust::tuple<real_t, real_t, int> &values) const
        {
            // unpack the tuple
            real_t target = values.get<0>();
            real_t output = values.get<1>();
            int outputIdx = values.get<2>();

            // check if we have to skip this value
            int patIdx = outputIdx / layerSize;
            if (patTypes[patIdx] == PATTYPE_NONE)
                return 0;

            // calculate the error
            real_t diff = target - output;
            return (diff * diff);
        }
    };

    struct ComputeMseWaveformGrad
    {
	real_t *preError;
	real_t *preData;
	real_t *realTargetData;

	int     preDim;
        int     featDim;
	real_t  beta;
	bool    accumulate;
	
        const char *patTypes;

        __host__ __device__ void operator() (const thrust::tuple<real_t, int> &values) const
        {
            // unpack the tuple
            int timeIdx = values.get<1>() / featDim;
	    int dimIdx  = values.get<1>() % featDim;

            if (patTypes[timeIdx] == PATTYPE_NONE){
		preError[timeIdx * preDim + dimIdx] = 0.0;
	    }else{
		if (accumulate)
		    preError[timeIdx * preDim + dimIdx] +=
			(preData[timeIdx * preDim + dimIdx] -
			 realTargetData[timeIdx * featDim + dimIdx]) * beta;
		else
		    preError[timeIdx * preDim + dimIdx] =
			(preData[timeIdx * preDim + dimIdx] -
			 realTargetData[timeIdx * featDim + dimIdx]) * beta;
	    }
        }
    };

    struct TimeDomainRemoveWaveformVoiced
    {
	real_t  f0TimeResolution;
	real_t *f0DataBuffer;

	real_t  f0DataM;
	real_t  f0DataS;
	
	int     f0InputLayerDim;
	int     waveformLength;
	int     featDim;
	
	const char *patTypes;
	
	__host__ __device__ void operator() (const thrust::tuple<real_t&, int> &values) const
	{
	    
            int timeIdx = values.get<1>() / featDim;
	    int f0TimeIdx = timeIdx / f0TimeResolution;
	    
            if (patTypes[timeIdx] == PATTYPE_NONE){
		values.get<0>() = 0.0;
	    }else{
		
		// remove voiced waveforms through time domain approach
		int distance = -1;
		real_t weight = 0.0;

		// if this point is in voiced frame
		if ((f0DataBuffer[f0TimeIdx * f0InputLayerDim + f0InputLayerDim - 1] *
		     f0DataS + f0DataM) > DFTERRORUV){

		    // first, look ahead, find the preceding unvoiced frame
		    for (int lookhead = 1;
			 (f0TimeIdx - lookhead) >=0 && lookhead <= DFTERRORUVSEARCH;
			 lookhead++){
			
			// if the frameframe is unvoiced
			//  make the boundary of unvoiced/voiced
			if ((f0DataBuffer[(f0TimeIdx - lookhead) * f0InputLayerDim +
					  f0InputLayerDim - 1] * f0DataS + f0DataM)
			    < DFTERRORUV){
			    distance = timeIdx - ((f0TimeIdx - lookhead + 1) * f0TimeResolution -1);
			    break;
			}
		    }

		    // second, look back, find the following unvoiced voices 
		    if (distance < 0){
			// first look ahead
			for (int lookback=1;
			     (f0TimeIdx + lookback) < waveformLength && lookback <= DFTERRORUVSEARCH;
			     lookback++){
			    if ((f0DataBuffer[(f0TimeIdx + lookback) * f0InputLayerDim +
					      f0InputLayerDim - 1] * f0DataS + f0DataM)
				< DFTERRORUV){
				distance = (f0TimeIdx + lookback) * f0TimeResolution - timeIdx;
				break;
			    }
			}	
		    }

		    if (distance < 0){
			// this time step is very inside a voiced frame, set it to zero directly
			values.get<0>() = 0.0;
		    }else{
			// if this time step is near the boundary of U/V change,
			//  set the voiced region based on Hann window
			weight = 0.5 * (1.0 + cos(2.0 * DFTERROR_PI * distance /
						  (2.0 * f0TimeResolution * DFTERRORUVSEARCH - 1)));
			values.get<0>() = weight * values.get<0>();
		    }
		}
	    }
	}
    };

    
    struct multiDimSignaltoOneDim
    {
	
	real_t *sourceData;
	int     sourceDim;
	int     maxLength;
	
        const char *patTypes;

        __host__ __device__ void operator() (const thrust::tuple<real_t&, int> &values) const
        {
            // unpack the tuple
            int dimIdx   = values.get<1>() / maxLength;
	    int timeIdx  = values.get<1>() % maxLength;

            if (patTypes[timeIdx] == PATTYPE_NONE){
		// dummy data point
		values.get<0>() = 0.0;
	    }else{
		values.get<0>() = sourceData[timeIdx * sourceDim + dimIdx];
	    }
        }
    };

    struct multiDimSignaltoOneDimGrad
    {
	
	real_t *sourceData;
	int     sourceDim;
	int     maxLength;
	
        const char *patTypes;

        __host__ __device__ void operator() (const thrust::tuple<real_t&, int> &values) const
        {
            // unpack the tuple
            int timeIdx = values.get<1>() / sourceDim;
	    int dimIdx  = values.get<1>() % sourceDim;

            if (patTypes[timeIdx] == PATTYPE_NONE){
		// dummy data point
		values.get<0>() = 0.0;
	    }else{
		values.get<0>() = sourceData[dimIdx * maxLength + timeIdx];
	    }
        }
    };

    struct preemphasis
    {
	
	real_t *sourceData;
	int     parallel;
	
        const char *patTypes;

        __host__ __device__ void operator() (const thrust::tuple<real_t&, int> &values) const
        {
            // unpack the tuple
            int timeIdx      = values.get<1>();
	    int blockIdx     = values.get<1>() / parallel;
	    int blockIntIdx  = values.get<1>() % parallel;

            if (patTypes[timeIdx] == PATTYPE_NONE){
		// dummy data point
		values.get<0>() = 0.0;
	    }else{
		// o[n] = s[n] - 0.94 * s[n-1]
		// o[n]: values.get<0>()
		// s[n]: sourceData[timeIdx]
		// s[n-1]: sourceData[(blockIdx-1) * parallel + blockIntIdx];
		
		if ((blockIdx - 1)>=0)
		    values.get<0>() = sourceData[timeIdx] -
			DFTPREEMPHASISCOEFF * sourceData[(blockIdx-1) * parallel + blockIntIdx];
		else
		    values.get<0>() = sourceData[timeIdx];
	    }
        }
    };

    struct deemphasis
    {
	
	real_t     *sourceData;
	int         parallel;
	int         maxWaveLength;
        const char *patTypes;

        __host__ __device__ void operator() (const thrust::tuple<real_t&, int> &values) const
        {
	    // de-emphasis is autoregressive, cannot be parallelized
	    // but can be parallelized cross different waveforms in the batch
	    
	    int blockIdx = values.get<1>();
	    for (int timeIdx = 0; timeIdx < maxWaveLength; timeIdx++){
		if (timeIdx > 0 && patTypes[timeIdx * parallel + blockIdx] != PATTYPE_NONE){
		    sourceData[timeIdx * parallel + blockIdx] =
			sourceData[timeIdx * parallel + blockIdx] +
			DFTPREEMPHASISCOEFF * sourceData[(timeIdx - 1) * parallel + blockIdx];
		}
		
	    }
        }
    };


    struct cleanImagPart
    {
	__host__ __device__ void operator() (complex_t &t) const
	{
	    t.y = 0;
	}
    };

    
}
}


namespace layers{

    // Construct the layer
    template <typename TDevice>
    DFTPostoutputLayer<TDevice>::DFTPostoutputLayer(
				const helpers::JsonValue &layerChild,
				Layer<TDevice> &precedingLayer,
				int maxSeqLength,
				int layerID)
        : PostOutputLayer<TDevice>(layerChild, precedingLayer,
				   precedingLayer.size(), maxSeqLength, layerID)
	, m_beta           (0.0)
	, m_gamma          (0.0)
	, m_zeta           (0.0)
	, m_mseError       (0.0)
	, m_noiseOutputLayer (NULL)
	, m_f0InputLayer     (NULL)
	, m_noiseTrain_epoch (-1)
	, m_modeMultiDimSignal (DFTMODEFORMULTIDIMSIGNAL_NONE)
    {
	
	if (precedingLayer.size() != this->size())
	    throw std::runtime_error("\nDFT Error previous layer size != DFT layer size");
		
	this->__loadOpts(layerChild);

	if (m_modeMultiDimSignal == DFTMODEFORMULTIDIMSIGNAL_NONE){
	    if (this->size() > 1){
		printf("\nError in dft layer: DFT layer size > 1, but multiDimSignalMode is off");
		throw std::runtime_error("\nError in network");
	    }
	    if (this->parallelSequences() > 1)
		throw std::runtime_error("\nDF TError is not implemented for parallel mode");
	}
    }	

    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::__cleanDFTError(struct_DFTData &dftBuf)
    {
	dftBuf.m_specError = 0.0;
	dftBuf.m_phaseError = 0.0;
	dftBuf.m_resError = 0.0;
	dftBuf.m_realSpecError = 0.0;
	dftBuf.m_lpcError = 0.0;
    }
    
    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::__initDFTBuffer(struct_DFTData &dftBuf)
    {
	dftBuf.m_valid_flag = false;
	
	this->__cleanDFTError(dftBuf);
	
	dftBuf.m_frameNum = 0;
	dftBuf.m_fftBinsNum = 0;
	dftBuf.m_frameLength = 0;
	dftBuf.m_frameShift = 0;
	dftBuf.m_windowType = 0;
	dftBuf.m_windowTypePhase = 0;
	dftBuf.m_fftLength = 0;

	dftBuf.m_fftLengthRealSpec = 0;
	dftBuf.m_fftBinsNumRealSpec = 0;

	dftBuf.m_lpcOrder = 0;
	
	dftBuf.m_fftSourceFramed.clear();
	dftBuf.m_fftTargetFramed.clear();
	dftBuf.m_fftSourceSigFFT.clear();
	dftBuf.m_fftTargetSigFFT.clear();
	
	dftBuf.m_fftDiffSigFFT.clear();
	dftBuf.m_fftDiffFramed.clear();
	dftBuf.m_fftDiffData.clear();
	dftBuf.m_fftDiffDataPhase.clear();
	dftBuf.m_fftResData.clear();
	
	dftBuf.m_fftSourceFramedRealSpec.clear();
	dftBuf.m_fftSourceSigFFTRealSpec.clear();
	dftBuf.m_fftTargetFramedRealSpec.clear();
	dftBuf.m_fftTargetSigFFTRealSpec.clear();
	
	dftBuf.m_fftDiffDataRealSpec.clear();
	dftBuf.m_fftDiffFramedRealSpec.clear();
	dftBuf.m_fftDiffSigFFTRealSpec.clear();

	dftBuf.m_autoCorrSrc.clear();
	dftBuf.m_lpcCoefSrc.clear();
	dftBuf.m_lpcErrSrc.clear();
	dftBuf.m_refCoefSrc.clear();
	
	dftBuf.m_autoCorrTar.clear();
	dftBuf.m_lpcCoefTar.clear();
	dftBuf.m_lpcErrTar.clear();
	dftBuf.m_refCoefTar.clear();

	dftBuf.m_lpcGrad.clear();
    }
    
    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::__configDFTBuffer(struct_DFTData &dftBuf,
							const int fftLength,
							const int frameLength,
							const int frameShift,
							const int windowType,
							const int windowTypePhase,
							const int lpcOrder)
    {
	
	//  make m_fftLength an even number
	int tmp_fftLength = (fftLength / 2) * 2;

	if (tmp_fftLength <= 0){
	    
	    // invalid DFT buffer
	    dftBuf.m_valid_flag = false;
	    return;
	    
	}else{
	    
	    // valid DFT buffer
	    dftBuf.m_valid_flag = true;

	    // configuration
	    dftBuf.m_fftLength = tmp_fftLength;
	    dftBuf.m_frameLength = frameLength;
	    dftBuf.m_frameShift = frameShift;
	    dftBuf.m_windowType = windowType;
	    dftBuf.m_lpcOrder = lpcOrder;
	    
	    // check
	    if (frameLength > tmp_fftLength)
		throw std::runtime_error("\nFFT length should be > frame length");

	    if (lpcOrder > frameLength)
		throw std::runtime_error("\nLPC order is too large");

	    // Maximum length of the waveforms in the training/test set
	    int maxSeqLength = this->__vMaxSeqLength();
	    
	    // number of fftBins (dimension of FFT feature) actually stored
	    dftBuf.m_fftBinsNum  = helpers::fftTools::fftBinsNum(dftBuf.m_fftLength);

	    // how many frames at maximum
	    dftBuf.m_frameNum = helpers::fftTools::fftFrameNum(maxSeqLength,
							       dftBuf.m_frameLength,
							       dftBuf.m_frameShift);

	    // allocate memory for FFT buffers
	    complex_t tmp;
	    tmp.x = 0;
	    tmp.y = 0;
	    
	    // framed signal buffer (with zero padding)
	    //   buffer for framed source signal
	    dftBuf.m_fftSourceFramed.resize(dftBuf.m_frameNum * dftBuf.m_fftLength, 0.0);
	    //   buffer for framed target signal
	    dftBuf.m_fftTargetFramed.resize(dftBuf.m_frameNum * dftBuf.m_fftLength, 0.0);
	    //   buffer for framed gradients
	    dftBuf.m_fftDiffFramed = dftBuf.m_fftTargetFramed;

	    //   buffer for STFT of source signal (complex number)
	    dftBuf.m_fftSourceSigFFT.resize(dftBuf.m_frameNum * dftBuf.m_fftBinsNum, tmp);
	    //   buffer for STFT of target signal
	    dftBuf.m_fftTargetSigFFT.resize(dftBuf.m_frameNum * dftBuf.m_fftBinsNum, tmp);
	    //   buffer for gradients (which is to be iSTFT into the 'framed gradients')
	    dftBuf.m_fftDiffSigFFT = dftBuf.m_fftTargetSigFFT;

	    // A temporary buffer to store the gradients
	    dftBuf.m_fftDiffData = this->outputs();

	    // gradient buffer phase distance
	    //  other data buffers are shared with specAmp distance
	    if (this->m_zeta > 0.0)
		dftBuf.m_fftDiffDataPhase = dftBuf.m_fftDiffData;

	    // buffer for complex-valued spectral distance
	    //  other data buffers are shared with specAmp distance	    
	    if (this->m_eta > 0.0)
		dftBuf.m_fftResData = dftBuf.m_fftDiffData;

	    // buffer for real-valued spectral distance
	    if (this->m_kappa > 0.0){
		dftBuf.m_fftLengthRealSpec  = dftBuf.m_fftLength * 2; 
		dftBuf.m_fftBinsNumRealSpec = helpers::fftTools::fftBinsNum(
				dftBuf.m_fftLengthRealSpec);
		dftBuf.m_fftSourceFramedRealSpec.resize(
				dftBuf.m_frameNum * dftBuf.m_fftLengthRealSpec, 0.0);
		dftBuf.m_fftTargetFramedRealSpec.resize(
				dftBuf.m_frameNum * dftBuf.m_fftLengthRealSpec, 0.0);
		dftBuf.m_fftDiffFramedRealSpec = dftBuf.m_fftTargetFramedRealSpec;
		dftBuf.m_fftSourceSigFFTRealSpec.resize(
				dftBuf.m_frameNum * dftBuf.m_fftBinsNumRealSpec, tmp);
		dftBuf.m_fftTargetSigFFTRealSpec.resize(
				dftBuf.m_frameNum * dftBuf.m_fftBinsNumRealSpec, tmp);
		dftBuf.m_fftDiffSigFFTRealSpec = dftBuf.m_fftTargetSigFFTRealSpec;
		dftBuf.m_fftDiffDataRealSpec   = this->outputs();
	    }

	    // buffer for LPC analysis
	    if (this->m_tau > 0.0){
		
		//if (dftBuf.m_lpcOrder == 0)
		// throw std::runtime_error("\nLPC order should be larger than 0");
		
		dftBuf.m_autoCorrSrc.resize(dftBuf.m_frameNum * (dftBuf.m_lpcOrder + 1), 0.0);
		dftBuf.m_lpcCoefSrc.resize(dftBuf.m_frameNum * (dftBuf.m_lpcOrder + 1) * 2, 0.0);
		dftBuf.m_lpcErrSrc.resize(dftBuf.m_frameNum * (dftBuf.m_lpcOrder + 1), 0.0);
		dftBuf.m_refCoefSrc.resize(dftBuf.m_frameNum * (dftBuf.m_lpcOrder + 1), 0.0);
		dftBuf.m_lpcResSrc.resize(dftBuf.m_frameNum * dftBuf.m_fftLength, 0.0);
		
		dftBuf.m_autoCorrTar = dftBuf.m_autoCorrSrc;
		dftBuf.m_lpcCoefTar = dftBuf.m_lpcCoefSrc;
		dftBuf.m_lpcErrTar  = dftBuf.m_lpcErrSrc;
		dftBuf.m_refCoefTar = dftBuf.m_refCoefSrc;
		dftBuf.m_lpcResTar = dftBuf.m_lpcResSrc;
		
		dftBuf.m_lpcGrad = dftBuf.m_fftDiffData;
		thrust::fill(dftBuf.m_lpcGrad.begin(), dftBuf.m_lpcGrad.end(), 0.0);
	    }
	}	
		
    }
    
    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::__loadOpts(const helpers::JsonValue &layerChild)
    {

	/* ------ Load general configuration ----- */
	
	/*
	  Error =  m_beta * waveform_MSE + m_gamma * spectral_amplitude_MSE + 
	           m_zeta * phase_MSE    + m_eta  * residual_signal_spectral_amplitude + 
		   m_kappa * real_spectrum_amp
	*/
	
	m_beta         = (layerChild->HasMember("beta") ? 
			   static_cast<real_t>((*layerChild)["beta"].GetDouble()) : 0.0);

	m_gamma        = (layerChild->HasMember("gamma") ? 
			   static_cast<real_t>((*layerChild)["gamma"].GetDouble()) : 0.0);

	m_zeta         = (layerChild->HasMember("zeta") ? 
			  static_cast<real_t>((*layerChild)["zeta"].GetDouble()) : 0.0);

	m_eta          = (layerChild->HasMember("eta") ? 
			  static_cast<real_t>((*layerChild)["eta"].GetDouble()) : 0.0);

	m_kappa        = (layerChild->HasMember("kappa") ? 
			  static_cast<real_t>((*layerChild)["kappa"].GetDouble()) : 0.0);
	
	m_tau        = (layerChild->HasMember("tau") ? 
			  static_cast<real_t>((*layerChild)["tau"].GetDouble()) : 0.0);
	
	// Type of spectral amplitude distance (see ../helpers/FFTMat.hpp):
	m_specDisType   = (layerChild->HasMember("specDisType") ? 
			   static_cast<real_t>((*layerChild)["specDisType"].GetInt()) :
			   FFTMAT_SPECTYPE_AMP_LOG_MSE);

	m_phaseDisType  = (layerChild->HasMember("phaseDisType") ? 
			   static_cast<real_t>((*layerChild)["phaseDisType"].GetInt()) :
			   FFTMAT_PHASETYPE_COS);
	// Type of real-valued spectrum
	m_realSpecType    = (layerChild->HasMember("kappa_realspec_type") ? 
			     ((*layerChild)["kappa_realspec_type"].GetInt()) :
			     FFTMAT_REALSPEC_TYPE_NORMAL);
	
	m_realSpecDisType = (layerChild->HasMember("realSpecDisType") ? 
			   static_cast<real_t>((*layerChild)["realSpecDisType"].GetInt()) :
			   FFTMAT_SPECTYPE_AMP_LOG_MSE);
	
	// Reserved option
	//  if the target signal is multi-dimensional, we can convert the multi-dimensional
	//  signal into one-dimensional signal as a waveform, then calcualte FFT distance
	m_modeMultiDimSignal = (layerChild->HasMember("multiDimSignalMode") ? 
				static_cast<int>((*layerChild)["multiDimSignalMode"].GetInt()) :
				DFTMODEFORMULTIDIMSIGNAL_NONE);

	// Reserved option
	//  If the natural waveform has to be pre-emphasized, we can use this option to
	//  do pre-emphasis, rather than preparing new training data
	//  Generated waveform will be de-emphasized in function computeForward()
	m_preEmphasis = (layerChild->HasMember("preEmphasisNaturalWav") ? 
			 static_cast<int>((*layerChild)["preEmphasisNaturalWav"].GetInt()) : 0);


	// LPC related configuration
	m_lpcErrorType = (layerChild->HasMember("lpcErrorType") ? 
			 static_cast<int>((*layerChild)["lpcErrorType"].GetInt()) :
			  SIGPROCESS_LPC_ERR_TYPE_WAV_MSE);

	
	/* ------ Load DFT configurations ----- */
	if (m_gamma > 0.0 || m_zeta > 0.0 || m_eta > 0.0 || m_kappa > 0.0 || m_tau > 0.0){

	    // -------- First DFT configuration ----
	    
	    // N-points FFT
	    int tmp_fftLength = (layerChild->HasMember("fftLength") ? 
				 static_cast<int>((*layerChild)["fftLength"].GetInt()) : 512);
	    
	    // actual length of one frame, which must be smaller than m_fftLength
	    int tmp_frameLength = (layerChild->HasMember("frameLength") ? 
				   static_cast<int>((*layerChild)["frameLength"].GetInt()) : 320);
	    
	    // frame shift
	    int tmp_frameShift = (layerChild->HasMember("frameShift") ? 
				  static_cast<int>((*layerChild)["frameShift"].GetInt()) : 80);
	    
	    // type of window for STFT framing (used to calculate spectral amplitude)
	    int tmp_windowType  = (layerChild->HasMember("windowType") ? 
				   static_cast<real_t>((*layerChild)["windowType"].GetInt()) :
				   FFTMAT_WINDOW_HANN);
	    
	    // type of window for STFT framing (used to calculate phase)
	    int tmp_windowTypePhase = layerChild->HasMember("windowTypePhase") ? 
		static_cast<real_t>((*layerChild)["windowTypePhase"].GetInt()):FFTMAT_WINDOW_HANN;

	    // Order of LPC (if configured)
	    int tmp_lpcOrder  = (layerChild->HasMember("lpcOrder") ? 
			  static_cast<int>((*layerChild)["lpcOrder"].GetInt()) : 0);

	    
	    struct_DFTData dftBuf_1;
	    this->__initDFTBuffer(dftBuf_1);
	    this->__configDFTBuffer(dftBuf_1, tmp_fftLength, tmp_frameLength, tmp_frameShift,
				    tmp_windowType, tmp_windowTypePhase, tmp_lpcOrder);
	    this->m_DFTDataBuf.push_back(dftBuf_1);

	    if (this->m_DFTDataBuf[0].m_valid_flag == false)
		throw std::runtime_error("\nFFT configuration error in DFT layer");
	    
	    // -------- Second DFT configuration ----
	    tmp_fftLength        = (layerChild->HasMember("fftLength2") ? 
				   static_cast<int>((*layerChild)["fftLength2"].GetInt()) : 0);
	    tmp_frameLength      = (layerChild->HasMember("frameLength2") ? 
				   static_cast<int>((*layerChild)["frameLength2"].GetInt()) : 80);
	    tmp_frameShift       = (layerChild->HasMember("frameShift2") ? 
				   static_cast<int>((*layerChild)["frameShift2"].GetInt()) : 40);
	    tmp_windowType       = (layerChild->HasMember("windowType2") ? 
				    static_cast<real_t>((*layerChild)["windowType2"].GetInt()) :
				    FFTMAT_WINDOW_HANN);
	    tmp_lpcOrder         = (layerChild->HasMember("lpcOrder2") ? 
				    static_cast<int>((*layerChild)["lpcOrder2"].GetInt()) : 0);
	    tmp_windowTypePhase  = layerChild->HasMember("windowTypePhase2") ? 
		static_cast<real_t>((*layerChild)["windowTypePhase2"].GetInt()) :
		FFTMAT_WINDOW_HANN;
	
	    struct_DFTData dftBuf_2;
	    this->__initDFTBuffer(dftBuf_2);
	    this->__configDFTBuffer(dftBuf_2, tmp_fftLength, tmp_frameLength, tmp_frameShift,
				    tmp_windowType, tmp_windowTypePhase, tmp_lpcOrder);
	    this->m_DFTDataBuf.push_back(dftBuf_2);
	    

	    
	    /* --------   third FFT ------------ */
	    tmp_fftLength        = (layerChild->HasMember("fftLength3") ? 
				   static_cast<int>((*layerChild)["fftLength3"].GetInt()) : 0);
	    tmp_frameLength      = (layerChild->HasMember("frameLength3") ? 
				   static_cast<int>((*layerChild)["frameLength3"].GetInt()) : 80);
	    tmp_frameShift       = (layerChild->HasMember("frameShift3") ? 
				   static_cast<int>((*layerChild)["frameShift3"].GetInt()) : 40);
	    tmp_windowType       = (layerChild->HasMember("windowType3") ? 
				    static_cast<real_t>((*layerChild)["windowType3"].GetInt()) :
				    FFTMAT_WINDOW_HANN);
	    tmp_lpcOrder         = (layerChild->HasMember("lpcOrder3") ? 
				    static_cast<int>((*layerChild)["lpcOrder3"].GetInt()) : 0);
	    tmp_windowTypePhase  = layerChild->HasMember("windowTypePhase3") ? 
		static_cast<real_t>((*layerChild)["windowTypePhase3"].GetInt()) :
		FFTMAT_WINDOW_HANN;
	    
	    struct_DFTData dftBuf_3;
	    this->__initDFTBuffer(dftBuf_3);
	    this->__configDFTBuffer(dftBuf_3, tmp_fftLength, tmp_frameLength, tmp_frameShift,
				    tmp_windowType, tmp_windowTypePhase, tmp_lpcOrder);
	    this->m_DFTDataBuf.push_back(dftBuf_3);

	    // Note: here we only used three DFT configurations
	    // If more DFT configurations are to be used,
	    // Please also modify FFTMat.cu getCuFFTHandle_fft to return multiple FFT handlers
	}
	

	// -------  Obsolete (not used anymore)
	// for a special training strategy on h-NSF
	m_hnm_flag   = (layerChild->HasMember("hnmMode") ? 
			static_cast<int>((*layerChild)["hnmMode"].GetInt()) : 0);
	if (m_hnm_flag > 0){
	    m_noiseTrain_epoch = (layerChild->HasMember("noisePartTrainEpochNum") ? 
		static_cast<int>((*layerChild)["noisePartTrainEpochNum"].GetInt()) : 15);
	    
	    m_noiseOutputLayerName = (layerChild->HasMember("noiseOutputLayerName") ? 
				      ((*layerChild)["noiseOutputLayerName"].GetString()) : "");
	    
	    m_f0InputLayerName = (layerChild->HasMember("f0InputLayerName") ? 
				  ((*layerChild)["f0InputLayerName"].GetString()) : "");
	    printf("\n\tDFT error layers turns on HNM model mode [%d]", m_hnm_flag);

	    m_f0DataM = (layerChild->HasMember("f0DataMean")?
			 static_cast<real_t>((*layerChild)["f0DataMean"].GetDouble()):0);
	    m_f0DataS = (layerChild->HasMember("f0DataStd")?
			 static_cast<real_t>((*layerChild)["f0DataStd"].GetDouble()):1);

	    const Configuration &config = Configuration::instance();
	    if (config.f0dataMean_signalgen() > 0)
		m_f0DataM = config.f0dataMean_signalgen();
	    if (config.f0dataStd_signalgen() > 0)
		m_f0DataS = config.f0dataStd_signalgen();
	    printf("\n\tDFT errir layers receives F0 mean-%f std-%f", m_f0DataM, m_f0DataS);
	}

	// multiple dimensional signal
	if (m_modeMultiDimSignal != DFTMODEFORMULTIDIMSIGNAL_NONE)
	    m_modeChangeDataBuf = this->outputs();
	else
	    m_modeChangeDataBuf.clear();

	// pre-emmphasis
	if (m_preEmphasis)
	    printf("\n\tNatural waveform will be pre-emphasis before evaluating.");

	// Done
    }

    
    // Destructor
    template <typename TDevice>
    DFTPostoutputLayer<TDevice>::~DFTPostoutputLayer()
    {
    }


    // Deemphasis during generation
    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::__deEmphasis(const int timeLength)
    {
	
	{{
	    internal::deemphasis fn1;
	    fn1.sourceData = helpers::getRawPointer(this->precedingLayer().outputs());
	    fn1.parallel   = this->parallelSequences();
	    fn1.patTypes   = helpers::getRawPointer(this->patTypes());
	    fn1.maxWaveLength = timeLength;
		
	    // use the buffer of outputErrors to store results
	    thrust::for_each(
		   thrust::make_zip_iterator(
			thrust::make_tuple(
				this->outputs().begin(),
				thrust::counting_iterator<int>(0))),
		   thrust::make_zip_iterator(
			thrust::make_tuple(
				this->outputs().begin()            + this->parallelSequences(),
				thrust::counting_iterator<int>(0)  + this->parallelSequences())),
		   fn1);
	}}
	return;
    }

    // Pre-emphasis during training
    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::__preEmphasis(const int timeLength)
    {
	// pre-emphasis the natural waveform before evaluating
	{{
		internal::preemphasis fn1;
		fn1.sourceData = helpers::getRawPointer(this->outputs());
		fn1.parallel   = this->parallelSequences();
		fn1.patTypes   = helpers::getRawPointer(this->patTypes());

		// dirty code: use the buffer of outputErrors to store results
		thrust::for_each(
		   thrust::make_zip_iterator(
			thrust::make_tuple(this->outputErrors().begin(),
					   thrust::counting_iterator<int>(0))),
		   thrust::make_zip_iterator(
			thrust::make_tuple(this->outputErrors().begin()       + timeLength,
					   thrust::counting_iterator<int>(0)  + timeLength)),
		   fn1);
		this->outputs() = this->outputErrors();
		thrust::fill(this->outputErrors().begin(), this->outputErrors().end(), 0.0);
	    }}
    }


    // Flatten the multi-dimensional signal into a one-dimensional signal
    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::__flattenMultiDimSignalForward(const int timeLength)
    {
	// If the output of previous layer is multi-dimensional signal,
	// convert the N * T input data matrix into a 1-dim signal of length NT.

	// convert target data
	{{
	      internal::multiDimSignaltoOneDim fn1;
	      fn1.sourceData  = helpers::getRawPointer(this->outputs());
	      fn1.patTypes    = helpers::getRawPointer(this->patTypes());
	      fn1.maxLength   = this->precedingLayer().curMaxSeqLength();
	      fn1.sourceDim   = this->precedingLayer().parallelSequences() * this->size();
		    
	      thrust::for_each(
		      thrust::make_zip_iterator(
		       thrust::make_tuple(
			 m_modeChangeDataBuf.begin(),
			 thrust::counting_iterator<int>(0))),
		      thrust::make_zip_iterator(
		       thrust::make_tuple(
			 m_modeChangeDataBuf.begin()        + timeLength,
			 thrust::counting_iterator<int>(0)  + timeLength)),
		      fn1);
	}}
	    
	thrust::copy(m_modeChangeDataBuf.begin(), m_modeChangeDataBuf.end(),
		     this->outputs().begin());

	// convert target data
	{{
	       internal::multiDimSignaltoOneDim fn1;
	       fn1.sourceData  = helpers::getRawPointer(this->precedingLayer().outputs());
	       fn1.patTypes    = helpers::getRawPointer(this->patTypes());
	       fn1.maxLength   = this->precedingLayer().curMaxSeqLength();
	       fn1.sourceDim   = this->precedingLayer().parallelSequences() * this->size();
		    
	       thrust::for_each(
		      thrust::make_zip_iterator(
		       thrust::make_tuple(
			 m_modeChangeDataBuf.begin(),
			 thrust::counting_iterator<int>(0))),
		      thrust::make_zip_iterator(
		       thrust::make_tuple(
			 m_modeChangeDataBuf.begin()        + timeLength,
			 thrust::counting_iterator<int>(0)  + timeLength)),
		      fn1);
	}}
	m_modeChangeDataBuf.swap(this->precedingLayer().outputs());
    }


    // Flatten the multi-dimensional signal into a one-dimensional signal
    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::__flattenMultiDimSignalBackward(const int timeLength)
    {

	// re-store the generated data
	m_modeChangeDataBuf.swap(this->precedingLayer().outputs());

	// gradient of multiDimSignaltoOneDim
	// convert target data
	{
	    internal::multiDimSignaltoOneDimGrad fn1;
	    fn1.sourceData  = helpers::getRawPointer(this->precedingLayer().outputErrors());
	    fn1.patTypes    = helpers::getRawPointer(this->patTypes());
	    fn1.maxLength   = this->precedingLayer().curMaxSeqLength();
	    fn1.sourceDim   = this->precedingLayer().parallelSequences() * this->size();
		    
	    thrust::for_each(
		      thrust::make_zip_iterator(
		       thrust::make_tuple(
			 m_modeChangeDataBuf.begin(),
			 thrust::counting_iterator<int>(0))),
		      thrust::make_zip_iterator(
		       thrust::make_tuple(
			 m_modeChangeDataBuf.begin()        + timeLength,
			 thrust::counting_iterator<int>(0)  + timeLength)),
		      fn1);
	}
		
	thrust::copy(m_modeChangeDataBuf.begin(), m_modeChangeDataBuf.end(),
		     this->precedingLayer().outputErrors().begin());
    }


    // Obsolete
    // A special training mode for HNM, not used anymore
    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::__hnmSpecialForward(const int timeLength, const int nnState)
    {
	// if this layer is valid for HNM special mode
	if (m_noiseOutputLayer && m_f0InputLayer &&
	    this->getCurrTrainingEpoch() < m_noiseTrain_epoch &&
	    nnState == NN_STATE_GAN_NOGAN_TRAIN) {
		
	    // remove the voiced part in target waveforms based on U/V infor
	    {{
		    internal::TimeDomainRemoveWaveformVoiced fn1;
		    fn1.f0TimeResolution = m_f0InputLayer->getResolution();
		    fn1.f0InputLayerDim  = m_f0InputLayer->size();
		    fn1.f0DataBuffer     = helpers::getRawPointer(m_f0InputLayer->outputs());
		    fn1.waveformLength   = timeLength;
		    fn1.f0DataM          = m_f0DataM;
		    fn1.f0DataS          = m_f0DataS;
		    fn1.featDim          = this->__vSize();
		    fn1.patTypes         = helpers::getRawPointer(this->patTypes());
		    
		    thrust::for_each(
		      thrust::make_zip_iterator(
		       thrust::make_tuple(
			 this->outputs().begin(),
			 thrust::counting_iterator<int>(0))),
		      thrust::make_zip_iterator(
		       thrust::make_tuple(
			 this->outputs().begin() + timeLength * this->__vSize(),
			 thrust::counting_iterator<int>(0)+timeLength*this->__vSize())),
		      fn1);
	     }}
		
	    // copy the noise output as the generated waveforms
	    //  (in order to calculate the error on the unvoiced regions only)
	    thrust::copy(m_noiseOutputLayer->outputs().begin(),
			 m_noiseOutputLayer->outputs().end(),
			 this->precedingLayer().outputs().begin());
	}
    }
    
    // Obsolete
    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::__hnmSpecialBackward(const int timeLength, const int nnState)
    {
	if (m_noiseOutputLayer && m_f0InputLayer &&
	    this->getCurrTrainingEpoch() < m_noiseTrain_epoch &&
	    nnState == NN_STATE_GAN_NOGAN_TRAIN) {
		
		// copy the gradients w.r.t noise component to the noise output layer
		thrust::copy(this->precedingLayer().outputErrors().begin(),
			     this->precedingLayer().outputErrors().end(),
			     m_noiseOutputLayer->outputErrors().begin());

		// Set the gradients w.r.t harmonic to zero (by setting this->precedingLayer())
		//  because m_noiseOutputLayer will be a skip-layer,
		//  this->precedingLayer send 0 to m_noiseOutputLayer.outputErrorFromSkipLayers,
		//  the gradients from DFT will be kept in m_noiseOutputLayer.outputErrors()
		thrust::fill(this->precedingLayer().outputErrors().begin(),
			     this->precedingLayer().outputErrors().end(),
			     0.0);
	    }

	    // for special mode 2
	    //  Harmonic and noise part are trained separatedly
	    //  When training on harmonic part, the gradients w.r.t noise component will be zero
	    if (m_noiseOutputLayer && m_f0InputLayer &&
		this->getCurrTrainingEpoch() >= m_noiseTrain_epoch &&
		nnState == NN_STATE_GAN_NOGAN_TRAIN && m_hnm_flag == DFTERRORPOST_HNM_MODEL_2){
		// copy the gradients w.r.t noise component to the noise output layer
		thrust::copy(this->precedingLayer().outputErrors().begin(),
			     this->precedingLayer().outputErrors().end(),
			     m_noiseOutputLayer->outputErrors().begin());

		// set the gradients w.r.t noise component to zero
		//  this is done by the simple trick below
		//  this trick cancles the gradients when outputSkipErrors + outputErrors for
		//  noise component
		thrust::negate<real_t> op;
		thrust::transform(m_noiseOutputLayer->outputErrors().begin(),
				  m_noiseOutputLayer->outputErrors().end(),
				  m_noiseOutputLayer->outputErrors().begin(),
				  op);
	    }
    }


    // Waveform MSE error
    template <typename TDevice>
    real_t DFTPostoutputLayer<TDevice>::__waveformMseForward(const int timeLength)
    {
	internal::ComputeMseWaveform fn;
	fn.layerSize = this->__vSize();
	fn.patTypes  = helpers::getRawPointer(this->patTypes());

	real_t mseError =
		(real_t)thrust::transform_reduce(
		   thrust::make_zip_iterator(
		      thrust::make_tuple(
			 this->outputs().begin(),
			 this->precedingLayer().outputs().begin(),   
			 thrust::counting_iterator<int>(0))),
		   thrust::make_zip_iterator(
		     thrust::make_tuple(
			 this->outputs().begin()                  + timeLength * this->__vSize(),
			 this->precedingLayer().outputs().begin() + timeLength * this->__vSize(),
			 thrust::counting_iterator<int>(0)        + timeLength * this->__vSize())),
		   fn, (real_t)0, thrust::plus<real_t>()) / timeLength;
	return mseError;
    }

    // Waveform MSE error gradients
    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::__waveformMseBackward(const int timeLength)
    {
	{{
	    internal::ComputeMseWaveformGrad fn2;
	    fn2.preData  = helpers::getRawPointer(this->precedingLayer().outputs());
	    fn2.realTargetData = helpers::getRawPointer(this->_targets());
	    fn2.preDim   = this->__vSize();
	    fn2.featDim  = this->__vSize();
	    fn2.beta     = m_beta;
	    fn2.patTypes = helpers::getRawPointer(this->patTypes());
	    
	    fn2.accumulate = false;
	    fn2.preError   = helpers::getRawPointer(this->precedingLayer().outputErrors());
		
	    
	    thrust::for_each(
		  thrust::make_zip_iterator(
		   thrust::make_tuple(
		     this->outputs().begin(),
		     thrust::counting_iterator<int>(0))),
		  thrust::make_zip_iterator(
		   thrust::make_tuple(
		     this->outputs().begin()            + timeLength * this->__vSize(),
		     thrust::counting_iterator<int>(0)  + timeLength * this->__vSize())),
		  fn2);
	}}	
    }


    template <typename TDevice>
    real_t DFTPostoutputLayer<TDevice>::__specAmpDistance(struct_DFTData &dftBuf,
							const int timeLength)
    {
	// FFT 
	// step0. build the data structure
	// FFTMat(raw_signal, framed_signal, fft_buffer,
	//        frame_length, frame_shift, window_type, N_points_FFT, N_points_FFT/2+1,
	//        num_of_frame, maximum_signal_length, this_signal_length,
	//        special_distance_type)
	helpers::FFTMat<TDevice> sourceSig(
			&this->_actualOutputs(), &dftBuf.m_fftSourceFramed,
			&dftBuf.m_fftSourceSigFFT,
			dftBuf.m_frameLength, dftBuf.m_frameShift,
			dftBuf.m_windowType, dftBuf.m_fftLength, dftBuf.m_fftBinsNum,
			dftBuf.m_frameNum, this->__vMaxSeqLength(), timeLength,
			this->m_specDisType);

	helpers::FFTMat<TDevice> targetSig(
			&this->_targets(), &dftBuf.m_fftTargetFramed,
			&dftBuf.m_fftTargetSigFFT,
			dftBuf.m_frameLength, dftBuf.m_frameShift,
			dftBuf.m_windowType, dftBuf.m_fftLength, dftBuf.m_fftBinsNum,
			dftBuf.m_frameNum, this->__vMaxSeqLength(), timeLength,
			this->m_specDisType);

	helpers::FFTMat<TDevice> fftDiffSig(
			&dftBuf.m_fftDiffData, &dftBuf.m_fftDiffFramed,
			&dftBuf.m_fftDiffSigFFT,
			dftBuf.m_frameLength, dftBuf.m_frameShift,
			dftBuf.m_windowType, dftBuf.m_fftLength, dftBuf.m_fftBinsNum,
			dftBuf.m_frameNum, this->__vMaxSeqLength(), timeLength,
			this->m_specDisType);
	
	// step1. framing and windowing
	sourceSig.frameSignal();
	targetSig.frameSignal();
		
	// step2. fft
	sourceSig.FFT();
	targetSig.FFT();

	// Dustbin #1
		
	// amplitude distance
	dftBuf.m_specError = sourceSig.specAmpDistance(targetSig, fftDiffSig);
	
	// compute complex-valued grad vector
	fftDiffSig.specAmpGrad(sourceSig, targetSig);
	// inverse DFT
	fftDiffSig.iFFT();
	// de-framing/windowing
	fftDiffSig.collectGrad(m_gamma);
	
	// Gradients should be in m_fftDiffData		    
	// Done
	return dftBuf.m_specError;
    }

    template <typename TDevice>
    real_t DFTPostoutputLayer<TDevice>::__specPhaDistance(struct_DFTData &dftBuf,
							const int timeLength)
    {
	// FFT 1
	// step0. build the data structure
	helpers::FFTMat<TDevice> sourceSigPhase(
			&this->_actualOutputs(), &dftBuf.m_fftSourceFramed,
			&dftBuf.m_fftSourceSigFFT,
			dftBuf.m_frameLength, dftBuf.m_frameShift,
			dftBuf.m_windowType, dftBuf.m_fftLength, dftBuf.m_fftBinsNum,
			dftBuf.m_frameNum, this->__vMaxSeqLength(), timeLength,
			this->m_phaseDisType);

	helpers::FFTMat<TDevice> targetSigPhase(
			&this->_targets(), &dftBuf.m_fftTargetFramed,
			&dftBuf.m_fftTargetSigFFT,
			dftBuf.m_frameLength, dftBuf.m_frameShift,
			dftBuf.m_windowType, dftBuf.m_fftLength, dftBuf.m_fftBinsNum,
			dftBuf.m_frameNum, this->__vMaxSeqLength(), timeLength,
			this->m_phaseDisType);

	helpers::FFTMat<TDevice> fftDiffSigPhase(
			&dftBuf.m_fftDiffDataPhase, &dftBuf.m_fftDiffFramed,
			&dftBuf.m_fftDiffSigFFT,
			dftBuf.m_frameLength, dftBuf.m_frameShift,
			dftBuf.m_windowType, dftBuf.m_fftLength, dftBuf.m_fftBinsNum,
			dftBuf.m_frameNum, this->__vMaxSeqLength(), timeLength,
			this->m_phaseDisType);
		
	// step1. framing and windowing
	sourceSigPhase.frameSignal();
	targetSigPhase.frameSignal();
		
	// step2. fft
	sourceSigPhase.FFT();
	targetSigPhase.FFT();

	// calculate phase distortion
	dftBuf.m_phaseError = sourceSigPhase.specPhaseDistance(targetSigPhase,
							       fftDiffSigPhase);
	// compute complex-valued grad vector
	fftDiffSigPhase.specPhaseGrad(sourceSigPhase, targetSigPhase);
	// inverse DFT
	fftDiffSigPhase.iFFT();
	// de-framing/windowing, grad will be in m_fftDiffDataPhase
	fftDiffSigPhase.collectGrad(m_zeta);

	return dftBuf.m_phaseError;
    }

    template <typename TDevice>
    real_t DFTPostoutputLayer<TDevice>::__specResAmpDistance(struct_DFTData &dftBuf,
							   const int timeLength)
    {
	// FFT 1	
	// step 1, calculate residual a = generated - target
	thrust::transform(this->_actualOutputs().begin(),
			  this->_actualOutputs().begin() + timeLength * this->__vSize(),
			  this->_targets().begin(),
			  dftBuf.m_fftResData.begin(),
			  thrust::minus<real_t>());
	    
	helpers::FFTMat<TDevice> fftResSig(
			&dftBuf.m_fftResData, &dftBuf.m_fftDiffFramed,
			&dftBuf.m_fftDiffSigFFT,
			dftBuf.m_frameLength, dftBuf.m_frameShift, dftBuf.m_windowType,
			dftBuf.m_fftLength, dftBuf.m_fftBinsNum,
			dftBuf.m_frameNum, this->__vMaxSeqLength(), timeLength,
			this->m_specDisType);
	
	// step 2. framing and windowing
	fftResSig.frameSignal();
	// step 3. fft
	fftResSig.FFT();		
	// amplitude distance
	dftBuf.m_resError = fftResSig.specAmpToZeroDistance();
	// compute complex-valued grad vector
	fftResSig.specAmpToZeroGrad();
	// inverse DFT
	fftResSig.iFFT();
	// de-framing/windowing
	fftResSig.collectGrad(m_eta);
	
	return dftBuf.m_resError;
    }

    template <typename TDevice>
    real_t DFTPostoutputLayer<TDevice>::__specRealAmpDistance(struct_DFTData &dftBuf,
							    const int timeLength)
    {
	
	helpers::FFTMat<TDevice> sourceSig(
			&this->_actualOutputs(), &dftBuf.m_fftSourceFramedRealSpec,
			&dftBuf.m_fftSourceSigFFTRealSpec,
			dftBuf.m_frameLength, dftBuf.m_frameShift, dftBuf.m_windowType,
			dftBuf.m_fftLengthRealSpec, dftBuf.m_fftBinsNumRealSpec,
			dftBuf.m_frameNum, this->__vMaxSeqLength(), timeLength,
			this->m_realSpecDisType);

	helpers::FFTMat<TDevice> targetSig(
			&this->_targets(), &dftBuf.m_fftTargetFramedRealSpec,
			&dftBuf.m_fftTargetSigFFTRealSpec,
			dftBuf.m_frameLength, dftBuf.m_frameShift, dftBuf.m_windowType,
			dftBuf.m_fftLengthRealSpec, dftBuf.m_fftBinsNumRealSpec,
			dftBuf.m_frameNum, this->__vMaxSeqLength(), timeLength,
			this->m_realSpecDisType);

	helpers::FFTMat<TDevice> fftDiffSig(
			&dftBuf.m_fftDiffDataRealSpec, &dftBuf.m_fftDiffFramedRealSpec,
			&dftBuf.m_fftDiffSigFFTRealSpec,
			dftBuf.m_frameLength, dftBuf.m_frameShift, dftBuf.m_windowType,
			dftBuf.m_fftLengthRealSpec, dftBuf.m_fftBinsNumRealSpec,
			dftBuf.m_frameNum, this->__vMaxSeqLength(), timeLength,
			this->m_realSpecDisType);
	
	// step1. framing and windowing
	sourceSig.frameSignalRealSpec(m_realSpecType);
	targetSig.frameSignalRealSpec(m_realSpecType);
		
	// step2. fft
	sourceSig.FFT();
	targetSig.FFT();

	// theoretically, imaginary part should be zero
	thrust::for_each(dftBuf.m_fftSourceSigFFTRealSpec.begin(),
			 dftBuf.m_fftSourceSigFFTRealSpec.end(),
			 internal::cleanImagPart());
	thrust::for_each(dftBuf.m_fftTargetSigFFTRealSpec.begin(),
			 dftBuf.m_fftTargetSigFFTRealSpec.end(),
			 internal::cleanImagPart());
	
	// amplitude distance
	dftBuf.m_realSpecError = sourceSig.specAmpDistance(targetSig, fftDiffSig);
	// compute complex-valued grad vector
	fftDiffSig.specAmpGrad(sourceSig, targetSig);
	// inverse DFT
	fftDiffSig.iFFT();
	// de-framing/windowing
	fftDiffSig.collectGradRealSpec(m_kappa, m_realSpecType, sourceSig);
	
	// Gradients should be in m_fftDiffData		    
	// Done
	return dftBuf.m_realSpecError;
    }


    template <typename TDevice>
    real_t DFTPostoutputLayer<TDevice>::__lpcError(struct_DFTData &dftBuf,
						 const int timeLength)
    {
	// number of frames for this utterance
	int validFrameNum =  helpers::fftTools::fftFrameNum(timeLength,
							    dftBuf.m_frameLength,
							    dftBuf.m_frameShift);
	// Warp the data structure
	// Assume m_fftSourceFramed and m_fftTargetFramed have stored the framed data.
	// (after __specAmpDistance())

	if (dftBuf.m_lpcOrder > 0){
	    helpers::lpcWarpper<TDevice> lpcAnalysizer(
		&dftBuf.m_fftSourceFramed, &dftBuf.m_fftTargetFramed,
		&dftBuf.m_autoCorrSrc, &dftBuf.m_autoCorrTar,
		&dftBuf.m_lpcCoefSrc, &dftBuf.m_lpcCoefTar,
		&dftBuf.m_lpcErrSrc, &dftBuf.m_lpcErrTar,
		&dftBuf.m_refCoefSrc, &dftBuf.m_refCoefTar,
		&dftBuf.m_lpcResSrc, &dftBuf.m_lpcResTar,
		&dftBuf.m_lpcGrad,
		dftBuf.m_lpcOrder,
		this->m_lpcErrorType,
		dftBuf.m_frameLength,
		dftBuf.m_frameShift,
		dftBuf.m_fftLength,
		validFrameNum,
		this->__vMaxSeqLength(), timeLength);

	    // Do LPC analysis
	    lpcAnalysizer.lpcAnalysis();

	    // Calculate LPC domain Errors
	    dftBuf.m_lpcError = lpcAnalysizer.lpcError();

	    // Collect gradients (which will be in dftBuf.m_lpcGrad)
	    lpcAnalysizer.lpcGradCollect(m_tau);
	    
	}else{
	    dftBuf.m_lpcError = 0.0;
	}
	
	return dftBuf.m_lpcError;
    }

    template <typename TDevice>
    real_t DFTPostoutputLayer<TDevice>::__specDistance_warpper(struct_DFTData &dftBuf,
							     const int timeLength)
    {
	real_t specAmpError = 0.0;
	real_t phaseError = 0.0;
	real_t specResError = 0.0;
	real_t specRealError = 0.0;
	real_t lpcError = 0.0;
	
	this->__cleanDFTError(dftBuf);
	
	// FFT configuration 1
	if (dftBuf.m_valid_flag){
		    
	    // calculate the spectral amplitude distance1
	    specAmpError = this->__specAmpDistance(dftBuf, timeLength);
	    
	    // calculate the phase distance
	    if (m_zeta > 0.0)
		phaseError = this->__specPhaDistance(dftBuf, timeLength);
	    
	    if (m_eta > 0.0)
		specResError = this->__specResAmpDistance(dftBuf, timeLength);
	    
	    if (m_kappa > 0.0)
		specRealError = this->__specRealAmpDistance(dftBuf, timeLength);

	    if (m_tau > 0.0)
		lpcError = this->__lpcError(dftBuf, timeLength);
	}
	return specAmpError + phaseError + specResError + specRealError + lpcError;
    }

    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::__specAccumulateGrad(struct_DFTData &dftBuf,
							   const int timeLength)
    {

	// Note: all the gradients have been computed in ComputeForwardPass
	// here, we only accumulate the gradients
	
	if (dftBuf.m_valid_flag){
	    
	    // grad of spec amplitude distance
	    thrust::transform(dftBuf.m_fftDiffData.begin(),
			      dftBuf.m_fftDiffData.begin() + timeLength * this->__vSize(),
			      this->precedingLayer().outputErrors().begin(),
			      this->precedingLayer().outputErrors().begin(),
			      thrust::plus<real_t>());
	    
	    // grad of phase amplitude distance
	    if (m_zeta > 0.0)
		thrust::transform(dftBuf.m_fftDiffDataPhase.begin(),
				  dftBuf.m_fftDiffDataPhase.begin() + timeLength * this->__vSize(),
				  this->precedingLayer().outputErrors().begin(),
				  this->precedingLayer().outputErrors().begin(),
				  thrust::plus<real_t>());

	    // grad of complex-valued spectral distance
	    if (m_eta > 0.0)
		thrust::transform(dftBuf.m_fftResData.begin(),
				  dftBuf.m_fftResData.begin() + timeLength * this->__vSize(),
				  this->precedingLayer().outputErrors().begin(),
				  this->precedingLayer().outputErrors().begin(),
				  thrust::plus<real_t>());

	    // grad of real-valued spectral distance
	    if (m_kappa > 0.0)
		thrust::transform(dftBuf.m_fftDiffDataRealSpec.begin(),
				  dftBuf.m_fftDiffDataRealSpec.begin() + timeLength*this->__vSize(),
				  this->precedingLayer().outputErrors().begin(),
				  this->precedingLayer().outputErrors().begin(),
				  thrust::plus<real_t>());

	    // grad of LPC error
	    if (m_tau > 0.0)
		thrust::transform(dftBuf.m_lpcGrad.begin(),
				  dftBuf.m_lpcGrad.begin() + timeLength * this->__vSize(),
				  this->precedingLayer().outputErrors().begin(),
				  this->precedingLayer().outputErrors().begin(),
				  thrust::plus<real_t>());
	}

    }
        
    
    // NN forward
    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::computeForwardPass(const int nnState)
    {
	// Checking 
	if (this->getSaveMemoryFlag())
	    throw std::runtime_error("Memory save mode should be turned off");

	// Length of the currennt waveform (or maximum length in the batch)
	int timeLength = this->__vCurMaxSeqLength();
		
	
	if (!this->flagTrainingMode()){
	    // Testing mode

	    // De-emphasis on the generated waveform
	    if (m_preEmphasis)
		this->__deEmphasis(timeLength);
	    
	    return;
	    
	}else{
	    
	    // Training mode
	    
	    // If preemphasis is necessary
	    if (m_preEmphasis)
		this->__preEmphasis(timeLength);
	    	    
	    // If target is multi-dimensional signal
	    if (m_modeMultiDimSignal == DFTMODEFORMULTIDIMSIGNAL_CONCATE)
		this->__flattenMultiDimSignalForward(timeLength);
	
	    // for HNM special training mode (obsolete)
	    if (m_hnm_flag == DFTERRORPOST_HNM_MODEL_1 || m_hnm_flag == DFTERRORPOST_HNM_MODEL_2)
		this->__hnmSpecialForward(timeLength, nnState);
	    	
	    // Compute waveform MSE if necessary 
	    if (m_beta > 0.0)
		m_mseError = this->__waveformMseForward(timeLength);
	    else
		m_mseError = 0.0;

	    // Compute DFT amplitute and phase distance if necessary
	    // This is the core part of this layer
	    // Note: these methods will
	    //   1. conduct signal_framing, STFT
	    //   2. calculate distances
	    //   3. calculate gradients and save them to the buffer
	    // 
	    if (m_gamma > 0.0 || m_zeta > 0.0 || m_eta > 0.0 || m_kappa > 0.0 || m_tau > 0.0){
		// calculate distances
		for (int dftBufIndex = 0; dftBufIndex < this->m_DFTDataBuf.size(); dftBufIndex++)
		    this->__specDistance_warpper(this->m_DFTDataBuf[dftBufIndex], timeLength);
	    }
	}
	return;
	// Done
    }

    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::computeForwardPass(const int timeStep, const int nnState)
    {
	// Not implemented yet
	//throw std::runtime_error("Not implemented DFTError computeForwardPass(timeStep)");
    }


    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::computeBackwardPass(const int nnState)
    {
	if (this->getSaveMemoryFlag())
	    throw std::runtime_error("Memory save mode should be turned off");

	// length of the training utterance
	int timeLength = this->__vCurMaxSeqLength();
	
	// initialize the gradients buffer
	thrust::fill(this->precedingLayer().outputErrors().begin(),
		     this->precedingLayer().outputErrors().end(), 0.0);
	    
	// Gradients from waveform MSE
	if (m_beta > 0.0)
	    this->__waveformMseBackward(timeLength);

	// Gradients from spectral amplitude and phase
	//  gradients have been calculated in computeForwardPass()
	//  here, gradients are simply accumulated into the gradient buffer
	if (m_gamma > 0.0 || m_zeta > 0.0 || m_eta > 0.0 || m_kappa > 0.0 || m_tau > 0.0){

	    // Accumulate gradients from each DFT buffer
	    for (int dftBufIndex = 0; dftBufIndex < this->m_DFTDataBuf.size(); dftBufIndex++)
		this->__specAccumulateGrad(this->m_DFTDataBuf[dftBufIndex], timeLength);
	}
	
	// For HNM special training mode
	if (m_hnm_flag == DFTERRORPOST_HNM_MODEL_1 || m_hnm_flag == DFTERRORPOST_HNM_MODEL_2)
	    this->__hnmSpecialBackward(timeLength, nnState);
	    
	// For multi-dim signal case
	if (m_modeMultiDimSignal == DFTMODEFORMULTIDIMSIGNAL_CONCATE)
	    this->__flattenMultiDimSignalBackward(timeLength);

	   
	// done
    }
	
    
    template <typename TDevice>
    const std::string& DFTPostoutputLayer<TDevice>::type() const
    {
	static std::string s1("dft");
        return s1;
    }

    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::exportLayer(const helpers::JsonValue &layersArray,
						  const helpers::JsonAllocator &allocator) const
    {
	PostOutputLayer<TDevice>::exportLayer(layersArray, allocator);
	    
	if (m_beta > 0.0)
	    (*layersArray)[layersArray->Size() - 1].AddMember("beta", m_beta, allocator);

	if (m_specDisType != FFTMAT_SPECTYPE_AMP_LOG_MSE)
	    (*layersArray)[layersArray->Size() - 1].AddMember("specDisType", m_specDisType,
							      allocator);
	if (m_phaseDisType != FFTMAT_PHASETYPE_COS)
	    (*layersArray)[layersArray->Size() - 1].AddMember("phaseDisType", m_phaseDisType,
							      allocator);
	if (m_preEmphasis)
	    (*layersArray)[layersArray->Size() - 1].AddMember("preEmphasisNaturalWav",
							      m_preEmphasis,
							      allocator);
	
	if (m_gamma > 0.0){
	    (*layersArray)[layersArray->Size() - 1].AddMember("gamma", m_gamma, allocator);
	    
	    if (m_zeta > 0.0)
		(*layersArray)[layersArray->Size() - 1].AddMember("zeta", m_zeta, allocator);
	    
	    if (m_eta > 0.0)
		(*layersArray)[layersArray->Size() - 1].AddMember("eta", m_eta, allocator);
	    
	    if (m_kappa > 0.0){
		(*layersArray)[layersArray->Size() - 1].AddMember("kappa", m_kappa, allocator);

		if (m_realSpecType != FFTMAT_REALSPEC_TYPE_NORMAL)
		    (*layersArray)[layersArray->Size() - 1].AddMember("kappa_realspec_type",
								      m_realSpecType, allocator);
		
		if (m_realSpecDisType != FFTMAT_SPECTYPE_AMP_LOG_MSE)
		    (*layersArray)[layersArray->Size() - 1].AddMember("realSpecDisType",
								      m_specDisType, allocator);
	    }

	    if (m_tau > 0.0){
		(*layersArray)[layersArray->Size() - 1].AddMember("tau", m_tau, allocator);
		(*layersArray)[layersArray->Size() - 1].AddMember("lpcErrorType", m_lpcErrorType,
								  allocator);
	    }
	    
	    for (int dftBufIndex = 0; dftBufIndex < this->m_DFTDataBuf.size(); dftBufIndex++){

		
		if (dftBufIndex == 0 && m_DFTDataBuf[dftBufIndex].m_valid_flag){
		    (*layersArray)[layersArray->Size() - 1].AddMember(
			"fftLength",
			m_DFTDataBuf[dftBufIndex].m_fftLength,
			allocator);
		    (*layersArray)[layersArray->Size() - 1].AddMember(
			"frameLength",
			m_DFTDataBuf[dftBufIndex].m_frameLength,
			allocator);
		    (*layersArray)[layersArray->Size() - 1].AddMember(
			"frameShift",
			m_DFTDataBuf[dftBufIndex].m_frameShift,
			allocator);
		    if (m_DFTDataBuf[dftBufIndex].m_windowType != FFTMAT_WINDOW_HANN)
			(*layersArray)[layersArray->Size() - 1].AddMember(
			 "windowType",
			 m_DFTDataBuf[dftBufIndex].m_windowType,
			 allocator);
		    if (m_DFTDataBuf[dftBufIndex].m_windowTypePhase != FFTMAT_WINDOW_HANN)
			(*layersArray)[layersArray->Size() - 1].AddMember(
			 "windowTypePhase",
			 m_DFTDataBuf[dftBufIndex].m_windowTypePhase,
			 allocator);
		    (*layersArray)[layersArray->Size() - 1].AddMember(
			"lpcOrder",
			m_DFTDataBuf[dftBufIndex].m_lpcOrder,
			allocator);
		}else if (dftBufIndex == 1 && m_DFTDataBuf[dftBufIndex].m_valid_flag){
		    (*layersArray)[layersArray->Size() - 1].AddMember(
			"fftLength2",
			m_DFTDataBuf[dftBufIndex].m_fftLength,
			allocator);
		    (*layersArray)[layersArray->Size() - 1].AddMember(
			"frameLength2",
			m_DFTDataBuf[dftBufIndex].m_frameLength,
			allocator);
		    (*layersArray)[layersArray->Size() - 1].AddMember(
			"frameShift2",
			m_DFTDataBuf[dftBufIndex].m_frameShift,
			allocator);
		    if (m_DFTDataBuf[dftBufIndex].m_windowType != FFTMAT_WINDOW_HANN)
			(*layersArray)[layersArray->Size() - 1].AddMember(
			 "windowType2",
			 m_DFTDataBuf[dftBufIndex].m_windowType,
			 allocator);
		    if (m_DFTDataBuf[dftBufIndex].m_windowTypePhase != FFTMAT_WINDOW_HANN)
			(*layersArray)[layersArray->Size() - 1].AddMember(
			 "windowTypePhase2",
			 m_DFTDataBuf[dftBufIndex].m_windowTypePhase,
			 allocator);
		    (*layersArray)[layersArray->Size() - 1].AddMember(
			"lpcOrder2",
			m_DFTDataBuf[dftBufIndex].m_lpcOrder,
			allocator);
		}else if (dftBufIndex == 2 && m_DFTDataBuf[dftBufIndex].m_valid_flag){
		    (*layersArray)[layersArray->Size() - 1].AddMember(
			"fftLength3",
			m_DFTDataBuf[dftBufIndex].m_fftLength,
			allocator);
		    (*layersArray)[layersArray->Size() - 1].AddMember(
			"frameLength3",
			m_DFTDataBuf[dftBufIndex].m_frameLength,
			allocator);
		    (*layersArray)[layersArray->Size() - 1].AddMember(
			"frameShift3",
			m_DFTDataBuf[dftBufIndex].m_frameShift,
			allocator);
		    if (m_DFTDataBuf[dftBufIndex].m_windowType != FFTMAT_WINDOW_HANN)
			(*layersArray)[layersArray->Size() - 1].AddMember(
			 "windowType3",
			 m_DFTDataBuf[dftBufIndex].m_windowType,
			 allocator);
		    if (m_DFTDataBuf[dftBufIndex].m_windowTypePhase != FFTMAT_WINDOW_HANN)
			(*layersArray)[layersArray->Size() - 1].AddMember(
			 "windowTypePhase3",
			 m_DFTDataBuf[dftBufIndex].m_windowTypePhase,
			 allocator);
		    (*layersArray)[layersArray->Size() - 1].AddMember(
			"lpcOrder3",
			m_DFTDataBuf[dftBufIndex].m_lpcOrder,
			allocator);
		}else{
		    // pass
		}
	    }
	    

	    
	    if (m_hnm_flag > 0 ){
		(*layersArray)[layersArray->Size() - 1].AddMember("hnmMode", m_hnm_flag,
								  allocator);
		(*layersArray)[layersArray->Size() - 1].AddMember("noisePartTrainEpochNum",
								  m_noiseTrain_epoch, allocator);
		(*layersArray)[layersArray->Size() - 1].AddMember("noiseOutputLayerName",
								  m_noiseOutputLayerName.c_str(),
								  allocator);
		(*layersArray)[layersArray->Size() - 1].AddMember("f0InputLayerName",
								  m_f0InputLayerName.c_str(),
								  allocator);
		(*layersArray)[layersArray->Size() - 1].AddMember("f0DataMean",
								  m_f0DataM, allocator);
		(*layersArray)[layersArray->Size() - 1].AddMember("f0DataStd",
								  m_f0DataS, allocator);
		
	    }

	    if (m_modeMultiDimSignal != DFTMODEFORMULTIDIMSIGNAL_NONE){
		(*layersArray)[layersArray->Size() - 1].AddMember("multiDimSignalMode",
								  m_modeMultiDimSignal, allocator);
	
	    }
	}   
    }
        
    template <typename TDevice>
    real_t DFTPostoutputLayer<TDevice>::calculateError()
    {
	// calculation has been down in forward pass
	// just print and return the results

	if (Configuration::instance().verboseLevel() == OP_VERBOSE_LEVEL_1){

	    for (int dftBufIndex = 0; dftBufIndex < this->m_DFTDataBuf.size(); dftBufIndex++)
		std::cerr << m_DFTDataBuf[dftBufIndex].m_specError << ", ";
	    
	    std::cerr << m_mseError << ", 0" << ", ";
	    if (m_zeta > 0.0){
		for (int dftBufIndex = 0; dftBufIndex < this->m_DFTDataBuf.size(); dftBufIndex++)
		    std::cerr << m_DFTDataBuf[dftBufIndex].m_phaseError << ", ";
	    }
	    if (m_eta > 0.0){
		for (int dftBufIndex = 0; dftBufIndex < this->m_DFTDataBuf.size(); dftBufIndex++)
		    std::cerr << m_DFTDataBuf[dftBufIndex].m_resError << ", ";
	    }
	    if (m_kappa > 0.0){
		for (int dftBufIndex = 0; dftBufIndex < this->m_DFTDataBuf.size(); dftBufIndex++)
		    std::cerr << m_DFTDataBuf[dftBufIndex].m_realSpecError << ", ";		
	    }
	    
	    if (m_tau > 0.0){
		for (int dftBufIndex = 0; dftBufIndex < this->m_DFTDataBuf.size(); dftBufIndex++)
		    std::cerr << m_DFTDataBuf[dftBufIndex].m_lpcError << ", ";		
	    }
	}

	real_t sum_error = 0.0;
	for (int dftBufIndex = 0; dftBufIndex < this->m_DFTDataBuf.size(); dftBufIndex++){
	    sum_error += m_DFTDataBuf[dftBufIndex].m_specError;
	    sum_error += m_DFTDataBuf[dftBufIndex].m_phaseError;
	    sum_error += m_DFTDataBuf[dftBufIndex].m_resError;
	    sum_error += m_DFTDataBuf[dftBufIndex].m_realSpecError;
	    sum_error += m_DFTDataBuf[dftBufIndex].m_lpcError;
	}
	return sum_error;
	    
    }

    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::linkTargetLayer(Layer<TDevice> &targetLayer)
    {
	// for HNM model, link the noise outputlayer and F0 input layer
	if (m_hnm_flag > 0){
	    // for noise output layer
	    if (targetLayer.name() == m_noiseOutputLayerName){
		// to be done: assert that it is a skip-layer
		if (targetLayer.type() != "skipini" && targetLayer.type() != "skipadd")
		    throw std::runtime_error("noiseOutputLayer is not a skipini/skipadd layer");
		// assert layer size
		if (targetLayer.size() != this->__vSize())
		    throw std::runtime_error("noiseOutputLayer layer size != DFT layer size");
		
		m_noiseOutputLayer = &targetLayer;
		printf("\n\tDFTlayer get noise output from %s",m_noiseOutputLayer->name().c_str());
	    }
	    
	    // for F0 input layer
	    if (targetLayer.name() == m_f0InputLayerName){
		m_f0InputLayer = &targetLayer;
		printf("\n\tDFT layer get F0 infor from %s", m_f0InputLayer->name().c_str());
		printf(", assume last dimension of its output as F0");
	    }
	}
	return;
    }

    template <typename TDevice>
    void DFTPostoutputLayer<TDevice>::computeBackwardPass(const int timeStep, const int nnState)
    {
	if (timeStep == this->curMaxSeqLength())
	    this->computeBackwardPass(nnState);
    }
    
    template <typename TDevice>
    int  DFTPostoutputLayer<TDevice>::__vSize()
    {
	if (m_modeMultiDimSignal == DFTMODEFORMULTIDIMSIGNAL_CONCATE)
	    return 1;
	else
	    return this->size();
    }

    template <typename TDevice>
    int  DFTPostoutputLayer<TDevice>::__vMaxSeqLength()
    {
	if (m_modeMultiDimSignal == DFTMODEFORMULTIDIMSIGNAL_CONCATE)
	    return this->maxSeqLength() * this->size() * this->parallelSequences();
	else
	    return this->maxSeqLength() * this->parallelSequences();
    }


    template <typename TDevice>
    int  DFTPostoutputLayer<TDevice>::__vCurMaxSeqLength()
    {
	if (m_modeMultiDimSignal == DFTMODEFORMULTIDIMSIGNAL_CONCATE)
	    return this->curMaxSeqLength() * this->size()  * this->parallelSequences();
	else
	    return this->curMaxSeqLength() * this->parallelSequences();
    } 
    
    template class DFTPostoutputLayer<Cpu>;
    template class DFTPostoutputLayer<Gpu>;
    
}


/* Dustbin #1
   // When phase and amplitude use the same FFTMats
		// -- phase part
		if (m_zeta > 0.0){
		// calculate phase distortion
		m_phaseError = sourceSig.specPhaseDistance(targetSig, fftDiffSig);
		// compute complex-valued grad vector
		fftDiffSig.specPhaseGrad(sourceSig, targetSig);
		// inverse DFT
		fftDiffSig.iFFT();
		// de-framing/windowing
		fftDiffSig.collectGrad(m_zeta);
		// copy the gradients to the phase grad buffer
		m_fftDiffDataPhase = m_fftDiffData;
		}else{
		m_phaseError = 0;
		}
	

  Dustbin #2
	    // -- phase part
		    if (m_zeta > 0.0){
		    m_phaseError2 = sourceSig2.specPhaseDistance(targetSig2, fftDiffSig2);
		    fftDiffSig2.specPhaseGrad(sourceSig2, targetSig2);
		    fftDiffSig2.iFFT();
		    fftDiffSig2.collectGrad(m_zeta);
		    m_fftDiffDataPhase2 = m_fftDiffData2;
		    }else{
		    m_phaseError2 = 0;
		    }

  Dustbin #3
	    
		    // -- phase part
		    if (m_zeta > 0.0){
		    m_phaseError3 = sourceSig3.specPhaseDistance(targetSig3, fftDiffSig3);
		    fftDiffSig3.specPhaseGrad(sourceSig3, targetSig3);
		    fftDiffSig3.iFFT();
		    fftDiffSig3.collectGrad(m_zeta);
		    m_fftDiffDataPhase3 = m_fftDiffData3;
		    }else{
		    m_phaseError3 = 0;
		    }
	
*/
